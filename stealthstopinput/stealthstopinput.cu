#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <vector>
#define _USE_MATH_DEFINES
#include <math.h>
#include "permutations.cuh"

#define NUM_OUTPUTS 2
#define NUM_PARTICLES 12
#define NUM_INPUTS (NUM_PARTICLES*3)
#define NUM_LEPTONS 2
#define NUM_JETS 10
#define NUM_JETS_CONV 4
#define datastring "D:/stealthstop/data/"

std::vector<float> convertFeatures(std::vector<float> inputs);
std::vector<float> convertConvolutionFeatures(std::vector<float> inputs);

int main() {
	std::string infname;
	std::string outfname;
	bool convertInputs = false;
	bool convolveInputs = false;
	std::cout << "Enter input file name: ";
	std::cin >> infname;
	std::cout << "Enter output file name: ";
	std::cin >> outfname;
	std::cout << "Convert inputs? ";
	std::cin >> convertInputs;
	if (convertInputs) {
		std::cout << "Convolve particles? ";
		std::cin >> convolveInputs;
	}

	std::ifstream infile(datastring + infname);
	std::ofstream totaloutfile(datastring + outfname);

	if (!infile.is_open()) {
		std::cout << "Couldn't find input file." << std::endl;
		system("pause");
		return 0;
	}

	std::string line;

	while (std::getline(infile, line)) {
		std::string tok;
		std::stringstream lss(line);

		std::vector<float> outputs(NUM_OUTPUTS);
		std::vector<float> features(NUM_INPUTS);
		for (size_t i = 0; i < NUM_OUTPUTS; i++) {
			lss >> outputs[i];
		}
		for (size_t i = 0; i < NUM_INPUTS; i++) {
			lss >> features[i];
		}

		if (convertInputs) {
			if (convolveInputs)
				features = convertConvolutionFeatures(features);
			else
				features = convertFeatures(features);
		}

		for (size_t i = 0; i < outputs.size(); i++)
			totaloutfile << outputs[i] << " ";
		for (size_t i = 0; i < features.size(); i++) {
			totaloutfile << features[i] << " ";
		}
		totaloutfile << std::endl;
	}
}

std::vector<float> convertFeatures(std::vector<float> inputs) {
	std::vector<float> convFeatures(2*NUM_PARTICLES + (NUM_PARTICLES*(NUM_PARTICLES-1)/2));

	for (size_t i = 0; i < NUM_PARTICLES; i++) {
		if (inputs[3 * i] != 0)
			convFeatures[2 * i] = std::log(inputs[3 * i]);	//pt
		else
			convFeatures[2 * i] = 0;
		convFeatures[2 * i + 1] = inputs[3 * i + 2];	//eta
	}

	//all phi combinations
	size_t combNum = 0;
	for (size_t i = 0; i < NUM_PARTICLES; i++) {
		for (size_t j = i + 1; j < NUM_PARTICLES; j++) {
			float firstPhi = inputs[3 * i + 1];
			float secondPhi = inputs[3 * j + 1];
			float convPhi = fabs(firstPhi - secondPhi);
			if (convPhi > M_PI)
				convPhi = 2 * M_PI - convPhi;
			if (firstPhi == 0 || secondPhi == 0)
				convPhi = 0;
			convFeatures[2 * NUM_PARTICLES + combNum] = convPhi;
			combNum++;
		}
	}

	return convFeatures;
}

std::vector<float> convertConvolutionFeatures(std::vector<float> inputs) {
	std::vector<float> convFeatures;

	PermutationArray permLepton = createPermutation(2);
	PermutationArray permJets;

	do { 
		permJets = createPermutation(4);
		do {
			for (size_t l = 0; l < permLepton.indices.size(); l++) {
				size_t lepNum = permLepton.indices[l];
				float pt = inputs[3 * l];
				if (pt == 0)
					convFeatures.push_back(0);
				else
					convFeatures.push_back(std::log(pt));
				convFeatures.push_back(inputs[3 * l + 1]);
				convFeatures.push_back(inputs[3 * l + 2]);
			}
			for (size_t j = 0; j < permJets.indices.size(); j++) {
				size_t jetNum = permJets.indices[j];
				float pt = inputs[3 * (NUM_LEPTONS + j)];
				if (pt == 0)
					convFeatures.push_back(0);
				else
					convFeatures.push_back(std::log(pt));
				convFeatures.push_back(inputs[3 * (NUM_LEPTONS + j) + 1]);
				convFeatures.push_back(inputs[3 * (NUM_LEPTONS + j) + 2]);
			}
		} while (iteratePermutation(&permJets));
	} while (iteratePermutation(&permLepton));

	return convFeatures;
}

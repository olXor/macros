#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <vector>
#define _USE_MATH_DEFINES
#include <math.h>
#include "lester_mt2_bisect.h"

#define NUM_PARTICLES 5
//#define NUM_INPUTS (NUM_PARTICLES*3)
#define NUM_INPUTS 17
#define datastring "D:/stopsearch/data/"

#define NUM_CAVEMAN_WEIGHTS 17

float getCavemanVar(std::vector<float> inputs);
std::vector<float> convertFeatures(std::vector<float> inputs);

int main() {
	std::string infname = "eventSaveBTagsignalTN_750-1_1968700";
	//std::string infname = "eventSaveBTagsignalWBMatched_380-200_1894470";
	//std::string infname = "eventSaveBTagsignalWBMatched_350-200_J3g200_1994132";
	//std::string infname = "eventSavebackground_J3g200_7625071";
	//std::string infname = "eventSaveBTagbackground_7625071";
	//std::string infname = "eventSavesignalTN_750-1_1870265";
	//std::string infname = "eventSaveBTagsignalTN_185-5_1300000";
	//std::string outfname = "convRazorTest_WBSignal";
	//std::string outfname = "convRazorTest_background";
	//std::string outfname = "convFixPhiBTagMT2_TNSignal_750-1_1968700";
	//std::string outfname = "convFixPhiBTag_background_J3g200_7625071";
	//std::string outfname = "convRelPhiBTag_background_7625071";
	//std::string outfname = "eventSaveBTagbackgroundMT2_7625071";
	std::string outfname = "eventSaveBTagsignalTNMT2_750-1_1968700";
	//std::string outfname = "convRelPhiBTag_TNSignal_185-5_1300000";
	//std::string outfname = "convFixPhiBTagMT2_background_7625071";

	bool convertInputs = true;
	/*
	std::cout << "Enter input file name: ";
	std::cin >> infname;
	std::cout << "Enter output file name: ";
	std::cin >> outfname;
	std::cout << "Convert inputs? ";
	std::cin >> convertInputs;
	*/

	std::ifstream infile(datastring + infname);
	std::ofstream totaloutfile(datastring + outfname);

	if (!infile.is_open()) {
		std::cout << "Couldn't find input file." << std::endl;
		system("pause");
		return 0;
	}

	std::string line;

	while (std::getline(infile, line)) {
		std::string tok;
		std::stringstream lss(line);

		if (line == "MISS" || line == "CUT") {
			totaloutfile << line << std::endl;
			continue;
		}
		std::vector<float> features(NUM_INPUTS);
		for (size_t i = 0; i < NUM_INPUTS; i++) {
			lss >> features[i];
		}

		if (convertInputs) {
			features = convertFeatures(features);
		}

		for (size_t i = 0; i < features.size(); i++) {
			totaloutfile << features[i] << " ";
		}
		totaloutfile << std::endl;
	}
}

//particles: met, l1, l2, j1, j2
std::vector<float> convertFeatures(std::vector<float> inputs) {
	//size_t numConvFeatures = 2 * NUM_PARTICLES + (NUM_PARTICLES*(NUM_PARTICLES - 1) / 2);
	//size_t numConvFeatures = 0;
	std::vector<float> convFeatures;

	/*
	bool addBTags = true;
	for (size_t i = 0; i < NUM_PARTICLES; i++) {
		size_t numExtraVars = (i > 3 && addBTags ? i - 3 : 0);	//b-tags
		if (inputs[3 * i + numExtraVars] != 0)
			convFeatures.push_back(std::log(inputs[3 * i + numExtraVars]));	//pt
		else
			convFeatures.push_back(0);
		//float missEtaSign = inputs[2] / fabs(inputs[2]);
		float lep1EtaSign = inputs[5] / fabs(inputs[5]);
		if (i != 0)
			convFeatures.push_back(inputs[3 * i + 2 + numExtraVars] / lep1EtaSign);	//eta

		if (i > 2 && addBTags)	//b-tag
			convFeatures.push_back(2 * inputs[3 * i + 3 + numExtraVars] - 1.0f);
	}

	float firstPhiSign = 1;
	//non-overlapping phi combinations
	for (size_t i = 1; i < NUM_PARTICLES; i++) {
		size_t numExtraVars = (i > 3 && addBTags ? i - 3 : 0);	//b-tags
		float firstPhi = inputs[1];	//MET
		float secondPhi = inputs[3 * i + 1 + numExtraVars];
		float phiDiff = secondPhi - firstPhi;
		float convPhi = fabs(firstPhi - secondPhi);
		if (phiDiff > M_PI)
			phiDiff -= 2 * M_PI;
		if (phiDiff < -M_PI)
			phiDiff += 2 * M_PI;
		float phiSign = (phiDiff > 0 ? 1 : -1);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
		if (firstPhi == 0 || secondPhi == 0)
			convPhi = 0;
		convFeatures.push_back(convPhi);
		if (i == 1)
			firstPhiSign = phiSign;
		else
			convFeatures.push_back(phiSign/firstPhiSign);
	}
	*/
	for (size_t i = 0; i < inputs.size(); i++)
		convFeatures.push_back(inputs[i]);

	/*
	//all phi combinations
	for (size_t i = 0; i < NUM_PARTICLES; i++) {
		for (size_t j = i + 1; j < NUM_PARTICLES; j++) {
			float firstPhi = inputs[3 * i + 1];
			float secondPhi = inputs[3 * j + 1];
			float convPhi = fabs(firstPhi - secondPhi);
			if (convPhi > M_PI)
				convPhi = 2 * M_PI - convPhi;
			if (firstPhi == 0 || secondPhi == 0)
				convPhi = 0;
			convFeatures.push_back(convPhi);
		}
	}
	*/

	//convFeatures.push_back(getCavemanVar(inputs));

	//convFeatures.push_back(inputs[0]);

	float metPt = inputs[0];
	float metPhi = inputs[1];
	float metPx = metPt*cos(metPhi);
	float metPy = metPt*sin(metPhi);
	float l1Pt = inputs[3];
	float l1Phi = inputs[4];
	float l1Eta = inputs[5];
	float l1Px = l1Pt*cos(l1Phi);
	float l1Py = l1Pt*sin(l1Phi);
	float l1Tanh = std::tanh(l1Eta);
	float l1Pz = l1Pt*l1Tanh / sqrt(1 - l1Tanh*l1Tanh);
	float l1E = sqrt(l1Pt*l1Pt + l1Pz*l1Pz);
	float l2Pt = inputs[6];
	float l2Phi = inputs[7];
	float l2Eta = inputs[8];
	float l2Px = l2Pt*cos(l2Phi);
	float l2Py = l2Pt*sin(l2Phi);
	float l2Tanh = std::tanh(l2Eta);
	float l2Pz = l2Pt*l2Tanh / sqrt(1 - l2Tanh*l2Tanh);
	float l2E = sqrt(l2Pt*l2Pt + l2Pz*l2Pz);
	float j1Pt = inputs[9];
	float j1Phi = inputs[10];
	float j1Eta = inputs[11];
	float j1Px = j1Pt*cos(j1Phi);
	float j1Py = j1Pt*sin(j1Phi);
	float j1Tanh = std::tanh(j1Eta);
	float j1Pz = j1Pt*j1Tanh / sqrt(1 - j1Tanh*j1Tanh);
	float j1E = sqrt(j1Pt*j1Pt + j1Pz*j1Pz);
	float j2Pt = inputs[12];
	float j2Phi = inputs[13];
	float j2Eta = inputs[14];
	float j2Px = j2Pt*cos(j2Phi);
	float j2Py = j2Pt*sin(j2Phi);
	float j2Tanh = std::tanh(j2Eta);
	float j2Pz = j2Pt*j2Tanh / sqrt(1 - j2Tanh*j2Tanh);
	float j2E = sqrt(j2Pt*j2Pt + j2Pz*j2Pz);

	//MT2(ll)
	float mt2ll = (float)asymm_mt2_lester_bisect::get_mT2(0, l1Px, l1Py, 0, l2Px, l2Py, metPx, metPy, 0, 0, 0.001);

	/*
	//MT2(blbl)
	float l1E = sqrt(l1Px*l1Px + l1Py*l1Py + l1Pz*l1Pz);
	float l2E = sqrt(l2Px*l2Px + l2Py*l2Py + l2Pz*l2Pz);
	float j1E = sqrt(j1Px*j1Px + j1Py*j1Py + j1Pz*j1Pz);
	float j2E = sqrt(j2Px*j2Px + j2Py*j2Py + j2Pz*j2Pz);
	float m1 = sqrt((l1E + j1E)*(l1E + j1E) - (l1Px + j1Px)*(l1Px + j1Px) - (l1Py + j1Py)*(l1Py + j1Py) - (l1Pz + j1Pz)*(l1Pz + j1Pz));
	float m2 = sqrt((l2E + j2E)*(l2E + j2E) - (l2Px + j2Px)*(l2Px + j2Px) - (l2Py + j2Py)*(l2Py + j2Py) - (l2Pz + j2Pz)*(l2Pz + j2Pz));
	//float mt2blbl = (float)asymm_mt2_lester_bisect::get_mT2(m1, l1Px + j1Px, l1Py + j1Py, m2, l2Px + j2Px, l2Py + j2Py, metPx, metPy, 0, 0, 0.001);
	float mt2blbl = (float)asymm_mt2_lester_bisect::get_mT2(0, l1Px + j1Px, l1Py + j1Py, 0, l2Px + j2Px, l2Py + j2Py, metPx, metPy, 0, 0, 0.001);

	//convFeatures.push_back(mt2blbl);
	*/

	//leptons only
	float q1Px = l1Px;
	float q1Py = l1Py;
	float q1Pz = l1Pz;
	float q1Pt = sqrt(pow(q1Px, 2) + pow(q1Py, 2));
	float q1E = l1E;
	float q2Px = l2Px;
	float q2Py = l2Py;
	float q2Pz = l2Pz;
	float q2Pt = sqrt(pow(q2Px, 2) + pow(q2Py, 2));
	float q2E = l2E;
	/*
	//megajets
	float q1Px = l1Px + j1Px;
	float q1Py = l1Py + j1Py;
	float q1Pz = l1Pz + j1Pz;
	float q1Pt = sqrt(pow(q1Px, 2) + pow(q1Py, 2));
	float q1E = l1E + j1E;
	float q2Px = l2Px + j2Px;
	float q2Py = l2Py + j2Py;
	float q2Pz = l2Pz + j2Pz;
	float q2Pt = sqrt(pow(q2Px, 2) + pow(q2Py, 2));
	float q2E = l2E + j2E;
	*/

	//Razor M_R
	float M_R = sqrt(pow(q1E + q2E, 2) - pow(q1Pz + q2Pz, 2));
	float M_TR = sqrt((metPt*(q1Pt + q2Pt) - metPx*(q1Px + q2Px) - metPy*(q1Py + q2Py)) / 2);
	float R_M2 = pow(M_TR / M_R, 2);

	//boost to longitudinal razor frame
	float beta_L = (q1Pz + q2Pz) / (q1E + q2E);
	float gamma_L = 1 / sqrt(1 - beta_L*beta_L);
	float q1E_L = gamma_L*(q1E - beta_L*q1Pz);
	float q1Px_L = q1Px;
	float q1Py_L = q1Py;
	float q1Pz_L = gamma_L*(q1Pz - beta_L*q1E);
	float q2E_L = gamma_L*(q2E - beta_L*q2Pz);
	float q2Px_L = q2Px;
	float q2Py_L = q2Py;
	float q2Pz_L = gamma_L*(q2Pz - beta_L*q2E);

	float m12sq = pow(q1E + q2E, 2) - pow(q1Px + q2Px, 2) - pow(q1Py + q2Py, 2) - pow(q1Pz + q2Pz, 2);
	float metE = sqrt(m12sq + metPx*metPx + metPy*metPy);

	//boost to super-razor frame (from L frame)
	float j_Tx = -metPx - q1Px - q2Px;
	float j_Ty = -metPy - q1Py - q2Py;
	float j_T = sqrt(j_Tx*j_Tx + j_Ty*j_Ty);
	float j_Tnx = j_Tx / j_T;
	float j_Tny = j_Ty / j_T;
	float s_R = 2 * (pow(M_R, 2) + j_Tx*(q1Px + q2Px) + j_Ty*(q1Py + q2Py) + M_R*sqrt(M_R*M_R + j_Tx*j_Tx + j_Ty*j_Ty + 2 * (j_Tx*(q1Px + q2Px) + j_Ty*(q1Py + q2Py))));
	float beta_R = -j_T / sqrt(s_R + pow(j_T, 2));
	float gamma_R = 1 / sqrt(1 - beta_R*beta_R);
	float q1E_R = gamma_R*(q1E_L - beta_R*q1Px_L*j_Tnx - beta_R*q1Py_L*j_Tny);
	float q1Px_R = ((1 + (gamma_R - 1)*j_Tnx*j_Tnx)*q1Px_L - beta_R*gamma_R*j_Tnx*q1E_L + (gamma_R - 1)*j_Tnx*j_Tny*q1Py_L);
	float q1Py_R = ((1 + (gamma_R - 1)*j_Tny*j_Tny)*q1Py_L - beta_R*gamma_R*j_Tny*q1E_L + (gamma_R - 1)*j_Tny*j_Tnx*q1Px_L);
	float q1Pz_R = q1Pz_L;
	float q2E_R = gamma_R*(q2E_L - beta_R*q2Px_L*j_Tnx - beta_R*q2Py_L*j_Tny);
	float q2Px_R = ((1 + (gamma_R - 1)*j_Tnx*j_Tnx)*q2Px_L - beta_R*gamma_R*j_Tnx*q2E_L + (gamma_R - 1)*j_Tnx*j_Tny*q2Py_L);
	float q2Py_R = ((1 + (gamma_R - 1)*j_Tny*j_Tny)*q2Py_L - beta_R*gamma_R*j_Tny*q2E_L + (gamma_R - 1)*j_Tny*j_Tnx*q2Px_L);
	float q2Pz_R = q2Pz_L;

	/*
	//boost to super-razor frame (from lab frame)
	float j_Tx = -metPx - q1Px - q2Px;
	float j_Ty = -metPy - q1Py - q2Py;
	float j_T = sqrt(j_Tx*j_Tx + j_Ty*j_Ty);
	float j_Tnx = j_Tx / j_T;
	float j_Tny = j_Ty / j_T;
	float s_R = 2 * (pow(M_R, 2) + j_Tx*(q1Px + q2Px) + j_Ty*(q1Py + q2Py) + M_R*sqrt(M_R*M_R + j_Tx*j_Tx + j_Ty*j_Ty + 2 * (j_Tx*(q1Px + q2Px) + j_Ty*(q1Py + q2Py))));
	float qSumPz = q1Pz + q2Pz;
	float pBoost = sqrt(j_T*j_T + qSumPz*qSumPz);
	float beta_R = pBoost / sqrt(pBoost*pBoost + s_R);
	float gamma_R = 1 / sqrt(1 - beta_R*beta_R);
	float nx_R = -j_Tx / pBoost;
	float ny_R = -j_Ty / pBoost;
	float nz_R = qSumPz / pBoost;
	float q1E_R = gamma_R*(q1E - beta_R*q1Px*nx_R - beta_R*q1Py*ny_R - beta_R*q1Pz*nz_R);
	float q1Px_R = (1 + (gamma_R - 1)*nx_R*nx_R)*q1Px - beta_R*gamma_R*nx_R*q1E + (gamma_R - 1)*nx_R*ny_R*q1Py + (gamma_R - 1)*nx_R*nz_R*q1Pz;
	float q1Py_R = (1 + (gamma_R - 1)*ny_R*ny_R)*q1Py - beta_R*gamma_R*ny_R*q1E + (gamma_R - 1)*ny_R*nx_R*q1Px + (gamma_R - 1)*ny_R*nz_R*q1Pz;
	float q1Pz_R = (1 + (gamma_R - 1)*nz_R*nz_R)*q1Pz - beta_R*gamma_R*nz_R*q1E + (gamma_R - 1)*nz_R*nx_R*q1Px + (gamma_R - 1)*nz_R*ny_R*q1Py;
	float q2E_R = gamma_R*(q2E - beta_R*q2Px*nx_R - beta_R*q2Py*ny_R - beta_R*q2Pz*nz_R);
	float q2Px_R = (1 + (gamma_R - 1)*nx_R*nx_R)*q2Px - beta_R*gamma_R*nx_R*q2E + (gamma_R - 1)*nx_R*ny_R*q2Py + (gamma_R - 1)*nx_R*nz_R*q2Pz;
	float q2Py_R = (1 + (gamma_R - 1)*ny_R*ny_R)*q2Py - beta_R*gamma_R*ny_R*q2E + (gamma_R - 1)*ny_R*nx_R*q2Px + (gamma_R - 1)*ny_R*nz_R*q2Pz;
	float q2Pz_R = (1 + (gamma_R - 1)*nz_R*nz_R)*q2Pz - beta_R*gamma_R*nz_R*q2E + (gamma_R - 1)*nz_R*nx_R*q2Px + (gamma_R - 1)*nz_R*ny_R*q2Py;
	*/

	//boost to decay frames
	float betaPx_R1 = (q1Px_R - q2Px_R) / (q1E_R + q2E_R);
	float betaPy_R1 = (q1Py_R - q2Py_R) / (q1E_R + q2E_R);
	float betaPz_R1 = (q1Pz_R - q2Pz_R) / (q1E_R + q2E_R);
	float beta_R1 = sqrt(pow(betaPx_R1, 2) + pow(betaPy_R1, 2) + pow(betaPz_R1, 2));
	float gamma_R1 = 1 / sqrt(1 - beta_R1*beta_R1);
	float M_deltaR = sqrt(s_R) / (2*gamma_R1);

	float q1dotM_L = q1E_L*metE - q1Px_L*metPx - q1Py_L*metPy;
	float q2dotM_L = q2E_L*metE - q2Px_L*metPx - q2Py_L*metPy;
	float M_deltaR_alt = sqrt((4 * q1dotM_L*q2dotM_L - pow(m12sq, 2)) / s_R);

	//R_pt
	float R_pt = j_T / (j_T + sqrt(s_R) / 4);

	//d_phiBR
	float qsumPx_R = q1Px_R + q2Px_R;
	float qsumPy_R = q1Py_R + q2Py_R;
	float qsumPz_R = q1Pz_R + q2Pz_R;
	float qsumPhi_R = atan2(qsumPy_R, qsumPx_R);
	float j_TPhi_R = atan2(-j_Ty, -j_Tx);
	float d_phiBR = fabs(j_TPhi_R - qsumPhi_R);
	if (d_phiBR > M_PI)
		d_phiBR = 2 * M_PI - d_phiBR;

	//cos \theta_(R+1) (note: based on megajets rather than leptons, if they are different)
	float costheta_R1 = sqrt(pow(q1E_R - q2E_R, 2) / (s_R / 4 - M_deltaR*M_deltaR));

	//m_delta
	float mS = 350;
	float mX = 200;
	float m_delta = (mS*mS - mX*mX) / mS;

	//cuts
	/*
	float feature = M_deltaR;
	if (R_pt <= 0.7)
		feature = 0;
	else if (1 / gamma_R1 <= 0.7)
		feature = 0;
	else if (d_phiBR <= 0.9*fabs(costheta_b) + 1.6)
		feature = 0;
		*/

	/*
	convFeatures.push_back(M_R);
	convFeatures.push_back(M_TR);
	convFeatures.push_back(R_M2);
	convFeatures.push_back(sqrt(s_R));
	*/
	//convFeatures.push_back(M_deltaR);
	//convFeatures.push_back(d_phiBR);

	convFeatures.push_back(mt2ll);
	return convFeatures;
}

float getCavemanVar(std::vector<float> inputs) {
	std::vector<float> weights = { 1.0f, 0.48f, 0.48f, 0.4f, 0.04f, 0.08f, 0.24f, 0.16f, 0.12f, 0.12f, 0.08f, 0, 0.04f, 0, 0.04f, -0.04f, 0.04f };
	std::vector<float> hardWeights(NUM_CAVEMAN_WEIGHTS);
	if (weights.size() != NUM_CAVEMAN_WEIGHTS)
		throw new std::runtime_error("Wrong number of caveman weights");

	float var = 0;
	var += weights[0] * inputs[0];	//MET
	var += weights[1] * inputs[11];	//MET-L2
	var += weights[2] * inputs[4];	//L2Pt
	var += weights[3] * inputs[10];	//MET-L1
	var += weights[4] * inputs[16];	//L1-J2
	var -= weights[5] * inputs[6];	//J1Pt
	var -= weights[6] * inputs[14];	//L1L2
	var += weights[7] * inputs[2];	//L1Pt
	var -= weights[8] * fabs(inputs[2] - 0.9*inputs[0]);	//L1Pt dependence on MET
	var -= weights[9] * inputs[18];	//L2-J2

	//L1Pt dependence on MET-L2
	if (inputs[11] < 0.7)
		var -= weights[10] * inputs[2];
	else if (inputs[11] < 1.07)
		var -= weights[11] * fabs(inputs[2] - (4 + 2 * (inputs[11] - 0.7) / 0.37));
	else if (inputs[11] < 1.238)
		var -= weights[10] * inputs[2];
	else if (inputs[11] < 2.18)
		var -= weights[12] * fabs(inputs[2] - (4 + 2 * (inputs[11] - 1.238) / (2.18 - 1.238)));
	else
		var += weights[13] * inputs[2];

	var -= weights[14] * inputs[12];	//MET-J1
	var += weights[15] * inputs[13];	//MET-J2
	if (inputs[2] < 5.3)	//MET-J2 dependence on L1Pt
		var -= weights[16] * fabs(inputs[13] - (2.33 + (3.14159 - 2.33)*(inputs[2] - 4.0f) / (6.0f - 4.0f)));

	return var;
}


#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <random>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

#define HEADER_SIZE 8
#define NUM_INPUTS 512
#define NUM_OUTPUTS 3

bool calculateAverage(std::string fname, float* average);
void saveMeanDiffs(std::string infname, std::string meanfname, std::string difffname, float mean);

int main() {
	std::string meanFolder;
	std::string diffFolder;
	std::cout << "Enter mean folder: ";
	std::cin >> meanFolder;
	std::cout << "Enter diff folder: ";
	std::cin >> diffFolder;

	std::ifstream filelist(datastring + (std::string)"filelist");
	std::ifstream indivlist(datastring + (std::string)"indivlist");

	std::vector<std::string> indivs;
	std::vector<float> indivAverages;
	std::vector<size_t> indivNumFiles;
	std::string line;
	while (std::getline(indivlist, line)) {
		indivs.push_back(line);
		indivAverages.push_back(0);
		indivNumFiles.push_back(0);
	}

	std::cout << "Computing individual averages: " << std::endl;
	while (std::getline(filelist, line)) {
		for (size_t i = 0; i < indivs.size(); i++) {
			if (line.substr(0, indivs[i].size()) == indivs[i]) {
				float average = 0;
				std::string fname;
				(std::stringstream(line) >> fname);
				if (!calculateAverage(fname, &average))
					break;
				indivAverages[i] += average;
				indivNumFiles[i]++;
				break;
			}
		}
	}

	for (size_t i = 0; i < indivs.size(); i++) {
		indivAverages[i] /= indivNumFiles[i];
		std::cout << indivs[i] << ": " << indivAverages[i] << " " << indivNumFiles[i] << std::endl;
	}

	std::cout << "Saving mean and differences: " << std::endl;
	filelist.clear();
	filelist.seekg(0, std::ios::beg);

	while (std::getline(filelist, line)) {
		for (size_t i = 0; i < indivs.size(); i++) {
			if (line.substr(0, indivs[i].size()) == indivs[i]) {
				std::string fname;
				(std::stringstream(line) >> fname);
				std::string localName = fname.substr(fname.find_last_of("\\/"), std::string::npos);
				saveMeanDiffs(fname, meanFolder + localName, diffFolder + localName, indivAverages[i]);
				break;
			}
		}
	}

	system("pause");
}

bool calculateAverage(std::string fname, float* average) {
	FILE* binfile = fopen(fname.c_str(), "rb");
	fseek(binfile, HEADER_SIZE, SEEK_SET);

	(*average) = 0;
	size_t numEvents = 0;
	std::vector<float> columns(NUM_INPUTS + NUM_OUTPUTS);
	while (fread(&columns[0], sizeof(float), NUM_INPUTS + NUM_OUTPUTS, binfile) == NUM_INPUTS + NUM_OUTPUTS) {
		(*average) += columns[0];
		numEvents++;
	}
	fclose(binfile);
	if (numEvents > 0)
		(*average) /= numEvents;
	return numEvents > 0;
}

void saveMeanDiffs(std::string infname, std::string meanfname, std::string difffname, float mean) {
	FILE* infile = fopen(infname.c_str(), "rb");
	FILE* meanfile = fopen(meanfname.c_str(), "wb");
	FILE* difffile = fopen(difffname.c_str(), "wb");
	char* header[HEADER_SIZE];
	fread(&header, HEADER_SIZE, 1, infile);
	fwrite(&header, HEADER_SIZE, 1, meanfile);
	fwrite(&header, HEADER_SIZE, 1, difffile);

	std::vector<float> columns(NUM_INPUTS + NUM_OUTPUTS);
	while (fread(&columns[0], sizeof(float), NUM_INPUTS + NUM_OUTPUTS, infile) == NUM_INPUTS + NUM_OUTPUTS) {
		columns[0] = columns[0] - mean;
		fwrite(&columns[0], sizeof(float), NUM_INPUTS + NUM_OUTPUTS, difffile);
		columns[0] = mean;
		fwrite(&columns[0], sizeof(float), NUM_INPUTS + NUM_OUTPUTS, meanfile);
	}

	fclose(infile);
	fclose(meanfile);
	fclose(difffile);
}
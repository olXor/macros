
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <sstream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <time.h>

#define CHECK_STEP_SIZE 100000

#define datastring "D:/stopSearch/data/"

void readInputs(std::vector<std::vector<float>>* inputs, std::string infname);
float median(float a, float b, float c);
void swap(std::vector<float>* A, size_t i, size_t j);
void quicksort(std::vector<float>* A, size_t lo, size_t hi);
void sortVector(std::vector<float>* A, size_t lo = 0, size_t hi = 0);
float evaluateMaxAMS(std::vector<std::vector<float>>* signalInputs, std::vector<std::vector<float>>* backgroundInputs, std::vector<std::vector<float>>* indivSort, std::vector<size_t>* cutLocs, std::vector<size_t>* maxCutLocs, size_t index, float stepSize);
void optimizeCuts(std::vector<std::vector<float>>* signalInputs, std::vector<std::vector<float>>* backgroundInputs, std::vector<std::vector<float>>* indivSort);
float evaluateAMS(std::vector<std::vector<float>>* signalInputs, std::vector<std::vector<float>>* backgroundInputs, std::vector<float> cuts);

int main() {
	srand((size_t)time(NULL));
	std::string signalFile;
	std::string backgroundFile;
	std::cout << "Enter signal file: ";
	std::cin >> signalFile;
	std::cout << "Enter background file: ";
	std::cin >> backgroundFile;

	bool manualCuts;
	std::cout << "Enter cuts manually? ";
	std::cin >> manualCuts;
	
	std::vector<std::vector<float>> signalInputs;
	std::vector<std::vector<float>> backgroundInputs;
	std::cout << "Reading inputs..." << std::endl;
	readInputs(&signalInputs, datastring + signalFile);
	readInputs(&backgroundInputs, datastring + backgroundFile);
	std::cout << "Sorting inputs..." << std::endl;
	std::vector<std::vector<float>> indivSort = signalInputs;
	for (size_t i = 0; i < backgroundInputs.size(); i++) {
		for (size_t j = 0; j < backgroundInputs[i].size(); j++)
			indivSort[i].push_back(backgroundInputs[i][j]);
	}
	for (size_t i = 0; i < indivSort.size(); i++) {
		sortVector(&indivSort[i]);
	}

	if (!manualCuts) {
		std::cout << "Optimizing..." << std::endl;
		for (size_t i = 0; i < 20; i++) {
			optimizeCuts(&signalInputs, &backgroundInputs, &indivSort);
			std::cout << std::endl;
		}
		std::cout << "Done. " << std::endl;
	}
	else {
		while (true) {
			std::vector<float> cuts(signalInputs.size());
			for (size_t i = 0; i < signalInputs.size(); i++) {
				std::cout << "Enter cut " << i << ": ";
				std::cin >> cuts[i];
			}
			std::cout << "AMS: " << evaluateAMS(&signalInputs, &backgroundInputs, cuts) << std::endl;
		}
	}
}

void optimizeCuts(std::vector<std::vector<float>>* signalInputs, std::vector<std::vector<float>>* backgroundInputs, std::vector<std::vector<float>>* indivSort) {
	std::vector<size_t> cutLocs;
	for (size_t i = 0; i < (*indivSort).size(); i++) {
		size_t randIndex = rand() + rand() * RAND_MAX;
		cutLocs.push_back(randIndex % (*indivSort)[i].size());
	}
	std::cout << "Starting Cuts: ";
	for (size_t i = 0; i < cutLocs.size(); i++)
		std::cout << (*indivSort)[i][cutLocs[i]] << " ";
	std::cout << std::endl;

	std::vector<size_t> maxCutLocs = cutLocs;
	float ams = 0;
	for (float stepSize = CHECK_STEP_SIZE; stepSize >= 1; stepSize *= 0.9) {
		do {
			cutLocs = maxCutLocs;
			ams = evaluateMaxAMS(signalInputs, backgroundInputs, indivSort, &cutLocs, &maxCutLocs, 0, stepSize);
		} while (cutLocs != maxCutLocs);
	}
	std::cout << "AMS: " << ams << " Cuts: ";
	for (size_t i = 0; i < maxCutLocs.size(); i++)
		std::cout << (*indivSort)[i][maxCutLocs[i]] << " ";
	std::cout << std::endl;
}

float evaluateMaxAMS(std::vector<std::vector<float>>* signalInputs, std::vector<std::vector<float>>* backgroundInputs, std::vector<std::vector<float>>* indivSort, std::vector<size_t>* cutLocs, std::vector<size_t>* maxCutLocs, size_t index, float stepSize) {
	if (index < cutLocs->size()) {
		float maxAMS = -9999;
		for (size_t diffRes = 0; diffRes < 3;diffRes++) {
			int diff = (diffRes == 2 ? -1 : (int)diffRes);
			std::vector<size_t> newCuts = (*cutLocs);
			std::vector<size_t> newMaxCuts = (*cutLocs);
			if (diff < 0 && size_t(newCuts[index] + diff*stepSize) >= (*indivSort)[index].size())
				newCuts[index] = 0;
			else if (diff > 0 && size_t(newCuts[index] + diff*stepSize) >= (*indivSort)[index].size())
				newCuts[index] = (*indivSort)[index].size() - 1;
			else
				newCuts[index] = (size_t)(newCuts[index] + diff*stepSize);
			float nextAMS = evaluateMaxAMS(signalInputs, backgroundInputs, indivSort, &newCuts, &newMaxCuts, index + 1, stepSize);
			if (nextAMS > maxAMS) {
				maxAMS = nextAMS;
				(*maxCutLocs) = newMaxCuts;
			}
		}
		return maxAMS;
	}

	(*maxCutLocs) = (*cutLocs);

	float srate = 0;
	float brate = 0;
	float breg = 10;
	for (size_t i = 0; i < (*signalInputs)[0].size(); i++) {
		bool chosen = true;
		for (size_t j = 0; j < (*signalInputs).size(); j++) {
			float cutVal = (*indivSort)[j][(*cutLocs)[j]];
			if ((*signalInputs)[j][i] <= cutVal) {
				chosen = false;
				break;
			}
		}

		if (chosen)
			srate += 0.00184 * 35900 / 1968700;	//35.9 fb^-1, 1.9e6 events
	}
	for (size_t i = 0; i < (*backgroundInputs)[0].size(); i++) {
		bool chosen = true;
		for (size_t j = 0; j < (*backgroundInputs).size(); j++) {
			float cutVal = (*indivSort)[j][(*cutLocs)[j]];
			if ((*backgroundInputs)[j][i] <= cutVal) {
				chosen = false;
				break;
			}
		}

		if (chosen)
			brate += 24.6 * 35900 / 7329772;	//7e6 events
	}

	float ams = std::sqrt(2 * ((srate + brate + breg)*std::log(1 + srate / (brate + breg)) - srate));
	return ams;
}

void readInputs(std::vector<std::vector<float>>* inputs, std::string infname) {
	std::ifstream infile(infname);
	if (!infile.is_open())
		std::cout << "Couldn't open input file " << infname << std::endl;
	inputs->clear();

	std::string line;
	while (std::getline(infile, line)) {
		std::stringstream lss(line);
		std::vector<float> ins;
		float val;
		while (lss >> val) {};	//take only last input

		ins.push_back(val);

		if (inputs->size() != 0 && inputs->size() != ins.size()) {
			std::cout << "Uneven number of inputs in file!" << std::endl;
			throw std::runtime_error("");
		}

		inputs->resize(ins.size());
		for (size_t i = 0; i < ins.size(); i++) {
			(*inputs)[i].push_back(ins[i]);
		}
	}
}

float median(float a, float b, float c) {
	return std::max(std::min(a, b), std::min(std::max(a, b), c));
}

void swap(std::vector<float>* A, size_t i, size_t j) {
	float tmp = (*A)[i];
	(*A)[i] = (*A)[j];
	(*A)[j] = tmp;
}

void quicksort(std::vector<float>* A, size_t lo, size_t hi) {
	if (lo >= hi)
		return;
	float pivot = median((*A)[lo], (*A)[hi], (*A)[(lo + hi) / 2]);
	size_t i = lo - 1;
	size_t j = hi + 1;
	while (true) {
		do {
			i++;
		} while (i <= hi && (*A)[i] < pivot);
		do{
			j--;
		} while (j >= lo && (*A)[j] > pivot);
		if (i >= j)
			break;
		swap(A, i, j);
	}
	quicksort(A, lo, j);
	quicksort(A, j + 1, hi);
}

void sortVector(std::vector<float>* A, size_t lo, size_t hi) {
	if (hi == 0)
		hi = A->size() - 1;
	quicksort(A, lo, hi);
}

float evaluateAMS(std::vector<std::vector<float>>* signalInputs, std::vector<std::vector<float>>* backgroundInputs, std::vector<float> cuts) {
	float srate = 0;
	float brate = 0;
	float breg = 10;
	for (size_t i = 0; i < (*signalInputs)[0].size(); i++) {
		bool chosen = true;
		for (size_t j = 0; j < (*signalInputs).size(); j++) {
			float cutVal = cuts[j];
			if ((*signalInputs)[j][i] <= cutVal) {
				chosen = false;
				break;
			}
		}

		if (chosen)
			srate += 0.00184 * 35900 / 1968700;	//35.9 fb^-1, 1.9e6 events
	}
	for (size_t i = 0; i < (*backgroundInputs)[0].size(); i++) {
		bool chosen = true;
		for (size_t j = 0; j < (*backgroundInputs).size(); j++) {
			float cutVal = cuts[j];
			if ((*backgroundInputs)[j][i] <= cutVal) {
				chosen = false;
				break;
			}
		}

		if (chosen)
			brate += 24.6 * 35900 / 7329772;	//7e6 events
	}

	std::cout << "srate: " << srate << " brate: " << brate << std::endl;
	float ams = std::sqrt(2 * ((srate + brate + breg)*std::log(1 + srate / (brate + breg)) - srate));
	return ams;
}
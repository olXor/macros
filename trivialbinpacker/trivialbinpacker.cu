#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include "sort.cuh"
#include "trivialbinpackerPeakFinder.cuh"

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

#define NUM_INPUTS 512
#define NUM_OUTPUTS 3//5
#define NUM_SILENT_OUTPUTS 0//12
#define HEADER_SIZE 8
#define TEST_FRACTION 1.0f
#define NUM_SAMPLES_PER_PERSON 2000//2000
#define NUM_VAL_SAMPLES_PER_PERSON 0
#define NUM_TEST_SAMPLES_PER_PERSON 2000//2000
#define MIN_SAMPLES_PER_PERSON 1//200
#define CUT_ON_AVERAGE_BP false//true
#define MIN_AVERAGE_OUTPUT 1//0
#define MAX_AVERAGE_OUTPUT 1000//0250
#define OUT_AVERAGE_SAMPLES 10000
#define OUT_AVERAGE_INDEX 1

#define UPSCALE_SAMPLES_PER_PERSON false

#define REMOVE_FLATLINES
#define FLATLINE_SEQ_POINTS 10

#define REPLACE_OLD_TESTFILES true

//#define NORMALIZE_BY_FIRST_TWO_OUTPUTS
#define NORMALIZE_BY_STDEV
//#define OUTLIERS_TO_TRAINSET
//#define APPLY_POST_TRANSFORM

#define USE_ALL_OUTPUTS true
#define SKIP_FIRST_OUTPUT false
#define USE_CALCULATED_OUTPUTS false
#define SAVE_OUTPUT 2
//#define PEAK_DATA_INCLUDED
#define NUM_SIDE_OUTPUT_PEAK_HEIGHTS 0	//if non-zero, overwrites normal outputs ifdef PEAK_DATA_INCLUDED

#define WAVEFORM_SMOOTHING_RANGE 0
//#define BASELINE_SHIFT_DISCARD_RANGE 5
#define BASELINE_SHIFT_DISCARD_THRESH 0.5
#define DERIV_SMOOTHING_RANGE 5

#define CONV_PEAK_FEATURES_REJECT_LOW_PEAK_WAVEFORMS
#define USE_EXTRA_CONV_PEAK_FEATURES 1
#define CONV_PEAK_FEATURES_INCLUDE_WAVEFORM true
#define CONV_PEAK_FEATURES_INCLUDE_ALL_SLOPES false
#define CONV_PEAK_FEATURES_INCLUDE_NEAR_SLOPE false
#define CONV_PEAK_FEATURES_INCLUDE_FAR_SLOPE false
#define CONV_PEAK_FEATURES_INCLUDE_X_POS false
#define CONV_PEAK_FEATURES_INCLUDE_Y_POS false
#define CONV_PEAK_FEATURES_INCLUDE_SLOPE_DIFF false
#define CONV_PEAK_FEATURES_INCLUDE_FIRST_DERIV false
#define CONV_PEAK_FEATURES_INCLUDE_SECOND_DERIV false
#define CONV_PEAK_FEATURES_INCLUDE_NORM_BY_CENTER false
#define CONV_PEAK_FEATURES_INCLUDE_ALL_DERIV2_SLOPES true

//#define USE_FIXED_FEATURES

size_t NUM_CV_SETS = 16;
size_t FIRST_CV = 0;

void readOldTestfiles(std::vector<std::vector<std::string>>* testfiles);
void createNewTestfiles(std::string filelist, std::vector<std::vector<std::string>>* testfiles);
size_t createSecondaryFeatures(float* inputs, float* peaks, std::vector<float>* secondaryFeatures, std::vector<bool>* localScaleMask);
void scaleConvSecondaryFeatures(std::vector<FILE*> trainsets, std::vector<FILE*> valsets, std::vector<FILE*> testsets, size_t numFeatures, std::vector<bool>* globalScaleMask);
size_t createSecondaryFixedFeatures(float* inputs, float* peaks, std::vector<float>* secondaryFeatures);
void calculatePeakOutputs(float* inputs, float* peaks, std::vector<float>* outputs);
void createScaleMasks(std::vector<bool>* globalScaleMask, std::vector<bool>* localScaleMask);

float transformVariable(float in) {
	float inup = (in + 1.0f)/2.0f;
	return (inup*inup*2 - 1.0f);
}

int main() {
	srand((size_t)time(NULL));

	std::cout << "Enter number of CV sets: ";
	std::cin >> NUM_CV_SETS;

	std::cout << "Enter first CV number: ";
	std::cin >> FIRST_CV;
	FIRST_CV--;

	bool useOldTestsets = false;
	std::cout << "Use old testfile lists? ";
	std::cin >> useOldTestsets;

	std::string filelist = "filelist";
	std::ifstream infilelist(datastring + filelist);
	if (!infilelist.is_open()) {
		std::cout << "Couldn't open file list " << datastring << filelist << std::endl;
	}

	std::vector<std::vector<std::string>> listtestfiles;
	if (useOldTestsets)
		readOldTestfiles(&listtestfiles);
	else {
		listtestfiles.resize(NUM_CV_SETS);
		createNewTestfiles(datastring + filelist, &listtestfiles);
	}

	std::string trainfname = "trainset";
	std::string testfname = "testset";
	std::string valfname = "valset";

	std::vector<std::ofstream> trainfilelists(NUM_CV_SETS);
	std::vector<std::ofstream> testfilelists(NUM_CV_SETS);

	std::vector<FILE*> trainsets(NUM_CV_SETS);
	std::vector<FILE*> valsets(NUM_CV_SETS);
	std::vector<FILE*> testsets(NUM_CV_SETS);
	size_t dum = 0;
	for (size_t i = 0; i < NUM_CV_SETS; i++) {
		std::stringstream numss;
		numss << "_" << FIRST_CV + i + 1;
		trainsets[i] = fopen((datastring + trainfname + numss.str()).c_str(), "wb+");
		fwrite(&dum, sizeof(size_t), 1, trainsets[i]);
		valsets[i] = fopen((datastring + valfname + numss.str()).c_str(), "wb+");
		fwrite(&dum, sizeof(size_t), 1, valsets[i]);
		testsets[i] = fopen((datastring + testfname + numss.str()).c_str(), "wb+");
		fwrite(&dum, sizeof(size_t), 1, testsets[i]);

		std::stringstream trainss;
		trainss << datastring << "trainfiles_" << FIRST_CV + i + 1;
		trainfilelists[i].open(trainss.str());
		if (!useOldTestsets || REPLACE_OLD_TESTFILES) {
			std::stringstream testss;
			testss << datastring << "testfiles_" << FIRST_CV + i + 1;
			testfilelists[i].open(testss.str());
		}
	}

	std::vector<size_t> trainSamples(NUM_CV_SETS);
	std::vector<size_t> valSamples(NUM_CV_SETS);
	std::vector<size_t> testSamples(NUM_CV_SETS);

#if USE_EXTRA_CONV_PEAK_FEATURES
	std::vector<bool> globalScaleMask;
	std::vector<bool> localScaleMask;
	createScaleMasks(&globalScaleMask, &localScaleMask);
#endif

	float personIdentifier = 0;
	std::string line;
	size_t numFeatures = 0;
	while (std::getline(infilelist, line)) {
		std::string fname;
		std::stringstream lss(line);
		lss >> fname;
		std::string groupName;
		if (!(lss >> groupName))
			groupName = fname;
		std::cout << "Reading file " << fname << ": ";

		size_t choiceNum;
		choiceNum = 0;
		for (size_t i = 0; i < listtestfiles.size(); i++) {
			for (size_t j = 0; j < listtestfiles[i].size(); j++) {
				if (listtestfiles[i][j] == groupName) {
					choiceNum = FIRST_CV + i + 1;
					break;
				}
			}
		}

		personIdentifier += 1.0f;

		for (size_t cv = 0; cv < NUM_CV_SETS; cv++) {
			if (FIRST_CV + cv + 1 == choiceNum) {
				if ((!useOldTestsets || REPLACE_OLD_TESTFILES)) {
					if (groupName == fname)
						testfilelists[cv] << personIdentifier << " " << fname << std::endl;
					else
						testfilelists[cv] << personIdentifier << " " << groupName << " " << fname << std::endl;
				}
			}
			else {
				if (groupName == fname)
					trainfilelists[cv] << personIdentifier << " " << fname << std::endl;
				else
					trainfilelists[cv] << personIdentifier << " " << groupName << " " << fname << std::endl;

			}
		}

		size_t* choiceSamples;

		FILE* infile = fopen((datastring + fname).c_str(), "rb");

		_fseeki64(infile, HEADER_SIZE, SEEK_SET);

		size_t numColumns = NUM_INPUTS + NUM_OUTPUTS + NUM_SILENT_OUTPUTS;
#ifdef PEAK_DATA_INCLUDED
		numColumns += NUM_INPUTS;
#endif

		std::vector<float> columns(numColumns);
		std::vector<size_t> sampleIndices;
		while (fread(&columns[0], sizeof(float), numColumns, infile) == numColumns) {
			sampleIndices.push_back(sampleIndices.size());
		}
		randomizeVector(&sampleIndices);

		std::cout << sampleIndices.size() << " samples; going to CV set" << choiceNum;
		if (sampleIndices.size() < MIN_SAMPLES_PER_PERSON) {
			std::cout << " Sample count under threshold; excluding" << std::endl;
			fclose(infile);
			continue;
		}

		float avgOutput = 0;
		size_t numAvgSamples = 0;
		for (size_t i = 0; i < std::min((size_t)OUT_AVERAGE_SAMPLES, sampleIndices.size()); i++) {
			_fseeki64(infile, sampleIndices[i]*numColumns*sizeof(float) + HEADER_SIZE, SEEK_SET);
			fread(&columns[0], sizeof(float), NUM_OUTPUTS, infile);
			avgOutput += columns[OUT_AVERAGE_INDEX - 1];
			numAvgSamples++;
		}
		avgOutput /= numAvgSamples;
		std::cout << " Average output: " << avgOutput << std::endl;
		if (CUT_ON_AVERAGE_BP && (avgOutput < MIN_AVERAGE_OUTPUT || avgOutput > MAX_AVERAGE_OUTPUT)) {
			std::cout << "Average output outside of accepted range. ";
#ifdef OUTLIERS_TO_TRAINSET
			choiceNum = 0;
			std::cout << "Going to all trainsets." << std::endl;
#else
			std::cout << "Ignoring. " << std::endl;
			fclose(infile);
			continue;
#endif
		}

		size_t trainSampleNum = NUM_SAMPLES_PER_PERSON + NUM_VAL_SAMPLES_PER_PERSON;
		size_t testSampleNum = (NUM_TEST_SAMPLES_PER_PERSON == 0 ? sampleIndices.size() : (size_t)NUM_TEST_SAMPLES_PER_PERSON);

		size_t numSamplesRemoved;
		size_t numTestSamplesRemoved = 0;

		for (size_t cv = 0; cv < NUM_CV_SETS; cv++) {
			FILE* dataset;
			size_t sampleNum;
			if (FIRST_CV + cv + 1 == choiceNum) {
				dataset = testsets[cv];
				choiceSamples = &testSamples[cv];
				sampleNum = testSampleNum;
				if (NUM_TEST_SAMPLES_PER_PERSON == 0) {
					for (size_t i = 0; i < sampleIndices.size(); i++)
						sampleIndices[i] = i;
				}
			}
			else {
				dataset = trainsets[cv];
				choiceSamples = &trainSamples[cv];
				sampleNum = trainSampleNum;
				randomizeVector(&sampleIndices);
			}

			numSamplesRemoved = 0;

			for (size_t i = 0; i < (UPSCALE_SAMPLES_PER_PERSON && !(FIRST_CV + cv + 1 == choiceNum && NUM_TEST_SAMPLES_PER_PERSON == 0) ? sampleNum + numSamplesRemoved : std::min(sampleNum + numSamplesRemoved, sampleIndices.size())); i++) {
				if (FIRST_CV + cv + 1 != choiceNum && i - numSamplesRemoved >= NUM_SAMPLES_PER_PERSON) {
					dataset = valsets[cv];
					choiceSamples = &valSamples[cv];
				}

				_fseeki64(infile, sampleIndices[i % sampleIndices.size()] * numColumns*sizeof(float) + HEADER_SIZE, SEEK_SET);
				fread(&columns[0], sizeof(float), numColumns, infile);

#ifdef PEAK_DATA_INCLUDED
				std::vector<float> calcOutputs;
				if (NUM_SIDE_OUTPUT_PEAK_HEIGHTS > 0) {
					calculatePeakOutputs(&columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS], &columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS + NUM_INPUTS], &calcOutputs);
				}
#endif

				float minInput = 9999;
				float maxInput = -9999;
#ifdef NORMALIZE_BY_FIRST_TWO_OUTPUTS
				minInput = columns[0];
				maxInput = columns[1];
#elif defined(NORMALIZE_BY_STDEV)
				float stdev = 0;
				float avg = 0;
				for (size_t in = 0; in < NUM_INPUTS; in++) {
					float input = columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS];
					stdev += input*input;
					avg += input;
				}
				stdev /= NUM_INPUTS;
				avg /= NUM_INPUTS;
				stdev = sqrt(stdev - avg*avg);

				for (size_t in = 0; in < NUM_INPUTS; in++) {
					columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS] = (columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS] - avg)/stdev;
				}
#else
				for (size_t in = 0; in < NUM_INPUTS; in++) {
					minInput = std::min(minInput, columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS]);
					maxInput = std::max(maxInput, columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS]);
				}
#endif

#ifndef NORMALIZE_BY_STDEV
				for (size_t in = 0; in < NUM_INPUTS; in++) {
					columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS] = 2.0f*(columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS] - minInput) / (maxInput - minInput) - 1.0f;
				}
#endif

#ifdef REMOVE_FLATLINES
				float lastVal = 9999;
				size_t flatSize = 0;
				for (size_t in = 0; in < NUM_INPUTS; in++) {
					float val = columns[in + NUM_OUTPUTS + NUM_SILENT_OUTPUTS];
					if (val == lastVal)
						flatSize++;
					else {
						flatSize = 0;
						lastVal = val;
					}
					if (flatSize > FLATLINE_SEQ_POINTS)
						break;
				}
				if (flatSize > FLATLINE_SEQ_POINTS) {
					numSamplesRemoved++;
					if (FIRST_CV + cv + 1 == choiceNum)
						numTestSamplesRemoved++;
					continue;
				}
#endif

#if defined(BASELINE_SHIFT_DISCARD_RANGE) && BASELINE_SHIFT_DISCARD_RANGE > 0
				float begAvg = 0;
				float endAvg = 0;
				float maxVal = -9999;
				float minVal = 9999;
				for (size_t in = 0; in < NUM_INPUTS; in++) {
					float val = columns[in + NUM_OUTPUTS];
					maxVal = std::max(maxVal, val);
					minVal = std::min(minVal, val);
					if(in < BASELINE_SHIFT_DISCARD_RANGE)
						begAvg += val;
					if(in > NUM_INPUTS - BASELINE_SHIFT_DISCARD_RANGE - 1)
						endAvg += val;
				}
				begAvg /= BASELINE_SHIFT_DISCARD_RANGE;
				endAvg /= BASELINE_SHIFT_DISCARD_RANGE;
				if (maxVal == minVal || fabs(endAvg - begAvg) / (maxVal - minVal) > BASELINE_SHIFT_DISCARD_THRESH) {
					numSamplesRemoved++;
					if (FIRST_CV + cv + 1 == choiceNum)
						numTestSamplesRemoved++;
					continue;
				}
#endif

#ifdef APPLY_POST_TRANSFORM
				for (size_t in = 0; in < NUM_INPUTS; in++) {
					columns[in + NUM_OUTPUTS] = transformVariable(columns[in + NUM_OUTPUTS]);
				}
#endif

#if defined(WAVEFORM_SMOOTHING_RANGE) && WAVEFORM_SMOOTHING_RANGE > 0
				std::vector<float> waveSmoothAverage(NUM_INPUTS);
				for (size_t in = 0; in < NUM_INPUTS; in++) {
					size_t count = 0;
					for (int j = -WAVEFORM_SMOOTHING_RANGE; j <= WAVEFORM_SMOOTHING_RANGE; j++) {
						if ((int)in + j >= 0 && (int)in + j < NUM_INPUTS) {
							count++;
							waveSmoothAverage[in] += columns[in + j + NUM_OUTPUTS];
						}
					}
					if (count > 0)
						waveSmoothAverage[in] /= count;
				}

				for (size_t in = 0; in < NUM_INPUTS; in++) {
					columns[in + NUM_OUTPUTS] = waveSmoothAverage[in];
				}
#endif

#if USE_EXTRA_CONV_PEAK_FEATURES
				std::vector<float> secondaryFeatures;
#ifdef PEAK_DATA_INCLUDED
				numFeatures = createSecondaryFeatures(&columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS], &columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS + NUM_INPUTS], &secondaryFeatures, &localScaleMask);
#else
				std::vector<float> waveform(NUM_INPUTS);
				for(size_t w=0;w<NUM_INPUTS;w++)
					waveform[w] = columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS + w];
				std::vector<float> genPeaks;
				findPeaksAndValleys(&waveform, &genPeaks);
				numFeatures = createSecondaryFeatures(&waveform[0], &genPeaks[0], &secondaryFeatures, &localScaleMask);
#endif
#elif defined(USE_FIXED_FEATURES)
				std::vector<float> secondaryFeatures;
				numFeatures = createSecondaryFixedFeatures(&columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS], &columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS + NUM_INPUTS], &secondaryFeatures);
#endif

				if (USE_ALL_OUTPUTS) {
					if (SKIP_FIRST_OUTPUT)
						fwrite(&columns[1], sizeof(float), NUM_OUTPUTS - 1, dataset);
					else
						fwrite(&columns[0], sizeof(float), NUM_OUTPUTS, dataset);
				}
#ifdef PEAK_DATA_INCLUDED
				else if (USE_CALCULATED_OUTPUTS && NUM_SIDE_OUTPUT_PEAK_HEIGHTS > 0) {
					fwrite(&calcOutputs[0], sizeof(float), calcOutputs.size(), dataset);
				}
#endif
				else
					fwrite(&columns[SAVE_OUTPUT - 1], sizeof(float), 1, dataset);
				fwrite(&personIdentifier, sizeof(float), 1, dataset);
#if USE_EXTRA_CONV_PEAK_FEATURES || defined(USE_FIXED_FEATURES)
				fwrite(&secondaryFeatures[0], sizeof(float), secondaryFeatures.size(), dataset);
#else
				fwrite(&columns[NUM_OUTPUTS + NUM_SILENT_OUTPUTS], sizeof(float), NUM_INPUTS, dataset);
#endif

				(*choiceSamples)++;
			}
		}
#if defined(REMOVE_FLATLINES) || (defined(BASELINE_SHIFT_DISCARD_RANGE) && BASELINE_SHIFT_DISCARD_RANGE > 0)
		std::cout << numTestSamplesRemoved << " samples discarded." << std::endl;
#endif
		fclose(infile);
	}
	for (size_t c = 0; c < NUM_CV_SETS; c++) {
		_fseeki64(trainsets[c], 0, SEEK_SET);
		fwrite(&trainSamples[c], sizeof(size_t), 1, trainsets[c]);

		_fseeki64(valsets[c], 0, SEEK_SET);
		fwrite(&valSamples[c], sizeof(size_t), 1, valsets[c]);

		_fseeki64(testsets[c], 0, SEEK_SET);
		fwrite(&testSamples[c], sizeof(size_t), 1, testsets[c]);
	}

#if USE_EXTRA_CONV_PEAK_FEATURES || defined(USE_FIXED_FEATURES)
	scaleConvSecondaryFeatures(trainsets, valsets, testsets, numFeatures, &globalScaleMask);
#endif
	
	for (size_t c = 0; c < NUM_CV_SETS; c++) {
		fclose(trainsets[c]);
		fclose(valsets[c]);
		fclose(testsets[c]);
	}

	std::cout << "Done. " << std::endl;
	std::cout << "Number of samples in first CV: trainset: " << trainSamples[0] << " testset: " << testSamples[0] << " valset: " << valSamples[0] << std::endl;

	system("pause");
}

void readOldTestfiles(std::vector<std::vector<std::string>>* testfiles) {
	testfiles->clear();
	for (size_t t = 0; t < NUM_CV_SETS; t++) {
		std::stringstream testss;
		testss << datastring << "testfiles_" << FIRST_CV + t + 1;
		std::ifstream testfile(testss.str());
		std::string line;
		std::vector<std::string> fnames;
		while (std::getline(testfile, line)) {
			size_t dum;
			std::string fname;
			std::stringstream lss(line);
			lss >> dum >> fname;
			if (std::find(fnames.begin(), fnames.end(), fname) == fnames.end())
				fnames.push_back(fname);
		}
		testfiles->push_back(fnames);
		testfile.close();
	}
}

void createNewTestfiles(std::string filelist, std::vector<std::vector<std::string>>* testfiles) {
	std::ifstream infile(filelist);
	std::vector<std::string> fnames;
	std::vector<size_t> ids;

	std::string line;
	size_t idNum = 0;
	while (std::getline(infile, line)) {
		std::stringstream lss(line);
		std::string idName;
		lss >> idName;
		std::string dum;
		if (lss >> dum)
			idName = dum;
		if (std::find(fnames.begin(), fnames.end(), idName) == fnames.end()) {
			fnames.push_back(idName);
			ids.push_back(idNum);
			idNum++;
		}
	}

	randomizeVector(&ids);

	for (size_t i = 0; i < testfiles->size(); i++) {
		std::vector<size_t> cvIds;
		for (size_t j = ids.size()*i / testfiles->size(); j < ids.size()*(i + 1) / testfiles->size(); j++)
			cvIds.push_back(ids[j]);

		(*testfiles)[i].clear();
		for (size_t j = 0; j < cvIds.size(); j++) {
			(*testfiles)[i].push_back(fnames[cvIds[j]]);
		}
	}
}

void createScaleMasks(std::vector<bool>* globalScaleMask, std::vector<bool>* localScaleMask) {
	globalScaleMask->clear();
	localScaleMask->clear();
	if (CONV_PEAK_FEATURES_INCLUDE_WAVEFORM) {
		globalScaleMask->push_back(false);
		localScaleMask->push_back(false);
	}

	if (CONV_PEAK_FEATURES_INCLUDE_ALL_SLOPES) {
		for (size_t i = 0; i < 4; i++) {
			globalScaleMask->push_back(true);
			localScaleMask->push_back(false);
		}
	}

	if (CONV_PEAK_FEATURES_INCLUDE_NEAR_SLOPE) {
		for (size_t i = 0; i < 2; i++) {
			globalScaleMask->push_back(true);
			localScaleMask->push_back(false);
		}
	}

	if (CONV_PEAK_FEATURES_INCLUDE_FAR_SLOPE) {
		for (size_t i = 0; i < 2; i++) {
			globalScaleMask->push_back(true);
			localScaleMask->push_back(false);
		}
	}

	if (CONV_PEAK_FEATURES_INCLUDE_X_POS) {
		for (size_t i = 0; i < 2; i++) {
			globalScaleMask->push_back(true);
			localScaleMask->push_back(false);
		}
	}

	if (CONV_PEAK_FEATURES_INCLUDE_Y_POS) {
		for (size_t i = 0; i < 2; i++) {
			globalScaleMask->push_back(true);
			localScaleMask->push_back(false);
		}
	}

	if (CONV_PEAK_FEATURES_INCLUDE_SLOPE_DIFF) {
		globalScaleMask->push_back(true);
		localScaleMask->push_back(false);
	}

	if (CONV_PEAK_FEATURES_INCLUDE_FIRST_DERIV) {
		globalScaleMask->push_back(false);
		localScaleMask->push_back(true);
	}

	if (CONV_PEAK_FEATURES_INCLUDE_SECOND_DERIV) {
		globalScaleMask->push_back(false);
		localScaleMask->push_back(true);
	}

	if (CONV_PEAK_FEATURES_INCLUDE_NORM_BY_CENTER) {
		globalScaleMask->push_back(false);
		localScaleMask->push_back(false);
	}

	if (CONV_PEAK_FEATURES_INCLUDE_ALL_DERIV2_SLOPES) {
		for (size_t i = 0; i < 4; i++) {
			globalScaleMask->push_back(true);
			localScaleMask->push_back(false);
		}
	}
}

size_t createSecondaryFeatures(float* inputs, float* peaks, std::vector<float>* secondaryFeatures, std::vector<bool>* localScaleMask) {
	size_t numFeatures = (CONV_PEAK_FEATURES_INCLUDE_ALL_SLOPES ? 4 : 0) + (CONV_PEAK_FEATURES_INCLUDE_NEAR_SLOPE ? 2 : 0) + (CONV_PEAK_FEATURES_INCLUDE_FAR_SLOPE ? 2 : 0) + (CONV_PEAK_FEATURES_INCLUDE_X_POS ? 2 : 0) + (CONV_PEAK_FEATURES_INCLUDE_Y_POS ? 2 : 0) + (CONV_PEAK_FEATURES_INCLUDE_WAVEFORM ? 1 : 0) + (CONV_PEAK_FEATURES_INCLUDE_SLOPE_DIFF ? 1 : 0) + (CONV_PEAK_FEATURES_INCLUDE_FIRST_DERIV ? 1 : 0) + (CONV_PEAK_FEATURES_INCLUDE_SECOND_DERIV ? 1 : 0) + (CONV_PEAK_FEATURES_INCLUDE_NORM_BY_CENTER ? 1 : 0) + (CONV_PEAK_FEATURES_INCLUDE_ALL_DERIV2_SLOPES ? 4 : 0);
	secondaryFeatures->clear();

	std::vector<size_t> peakLocs;
	std::vector<size_t> valleyLocs;
	float minVal = 9999;
	float maxVal = -9999;

	for (size_t i = 0; i < NUM_INPUTS; i++) {
		if (peaks[i] > 0)
			peakLocs.push_back(i);
		else if (peaks[i] < 0)
			valleyLocs.push_back(i);
		minVal = std::min(minVal, inputs[i]);
		maxVal = std::max(maxVal, inputs[i]);
	}
	float minPeakDistance = 9999;
	float minValleyDistance = 9999;
	size_t centerPeak = 0;
	size_t centerValley = 0;
	for (size_t i = 0; i < peakLocs.size(); i++) {
		int dist = abs((int)peakLocs[i] - NUM_INPUTS / 2);
		if (dist < minPeakDistance) {
			minPeakDistance = dist;
			centerPeak = peakLocs[i];
		}
	}
	for (size_t i = 0; i < valleyLocs.size(); i++) {
		int dist = abs((int)valleyLocs[i] - NUM_INPUTS / 2);
		if (dist < minValleyDistance) {
			minValleyDistance = dist;
			centerValley = valleyLocs[i];
		}
	}

#ifdef CONV_PEAK_FEATURES_REJECT_LOW_PEAK_WAVEFORMS
	if (peakLocs.size() < 2 || valleyLocs.size() < 2) {
		/*
		std::cout << "Found pulse with less than 2 peaks or valleys detected" << std::endl;
		for (size_t i = 0; i < NUM_INPUTS; i++) {
			std::cout << inputs[i] << " ";
		}
		std::cout << std::endl;
		for (size_t i = 0; i < NUM_INPUTS; i++) {
			std::cout << peaks[i] << " ";
		}
		std::cout << std::endl;
		*/
		for (size_t i = 0; i < NUM_INPUTS; i++) {
			for (size_t f = 0; f < numFeatures; f++) {
				(*secondaryFeatures).push_back(0);
			}
		}
		return numFeatures;
	}
#endif

	std::vector<float> waveform(NUM_INPUTS);
	std::vector<float> deriv1;
	std::vector<float> deriv2;

	for (size_t w = 0; w < NUM_INPUTS; w++)
		waveform[w] = inputs[w];

	if (CONV_PEAK_FEATURES_INCLUDE_FIRST_DERIV || CONV_PEAK_FEATURES_INCLUDE_SECOND_DERIV || CONV_PEAK_FEATURES_INCLUDE_ALL_DERIV2_SLOPES) {
		computeDerivative(&waveform, &deriv1, DERIV_SMOOTHING_RANGE);
	}

	if (CONV_PEAK_FEATURES_INCLUDE_SECOND_DERIV || CONV_PEAK_FEATURES_INCLUDE_ALL_DERIV2_SLOPES) {
		computeDerivative(&deriv1, &deriv2, DERIV_SMOOTHING_RANGE);
	}

	float maxDeriv1 = -99999;
	float minDeriv1 = 99999;
	float maxDeriv2 = -99999;
	float minDeriv2 = 99999;

	for (size_t i = 0; i < NUM_INPUTS; i++) {
		minDeriv1 = std::min(minDeriv1, deriv1[i]);
		maxDeriv1 = std::max(maxDeriv1, deriv1[i]);
		minDeriv2 = std::min(minDeriv2, deriv2[i]);
		maxDeriv2 = std::max(maxDeriv2, deriv2[i]);
	}

	size_t lastPeak = 0;
	size_t lastValley = 0;

	for (size_t i = 0; i < NUM_INPUTS; i++) {
		bool hasBackPeak = i > peakLocs[0];
		bool hasForwardPeak = i < peakLocs[peakLocs.size() - 1];
		bool hasBackValley = i > valleyLocs[0];
		bool hasForwardValley = i < valleyLocs[valleyLocs.size() - 1];

		if (lastPeak < peakLocs.size() - 1 && i > peakLocs[lastPeak + 1])
			lastPeak++;
		if (lastValley < valleyLocs.size() - 1 && i > valleyLocs[lastValley + 1])
			lastValley++;

		float pulseWidth = (lastPeak < peakLocs.size() - 1 ? 1.0f*(peakLocs[lastPeak + 1] - peakLocs[lastPeak]) : 1.0f*(peakLocs[lastPeak] - peakLocs[lastPeak - 1]));
		float peakXDiffBack;
		float peakYDiffBack;
		if (i > peakLocs[0]) {
			peakXDiffBack = 1.0f*(i - peakLocs[lastPeak]) / pulseWidth;
			peakYDiffBack = (maxVal > minVal ? (inputs[i] - inputs[peakLocs[lastPeak]]) / (maxVal - minVal) : 0.0f);
		}
		else {
			peakXDiffBack = 0;
			peakYDiffBack = 0;
		}

		float peakXDiffFor;
		float peakYDiffFor;
		if (i < peakLocs[peakLocs.size() - 1]) {
			peakXDiffFor = 1.0f*(peakLocs[lastPeak + 1] - i) / pulseWidth;
			peakYDiffFor = (maxVal > minVal ? (inputs[peakLocs[lastPeak + 1]] - inputs[i]) / (maxVal - minVal) : 0.0f);
		}
		else {
			peakXDiffFor = 0;
			peakYDiffFor = 0;
		}

		float valleyXDiffBack;
		float valleyYDiffBack;
		if (i > valleyLocs[0]) {
			valleyXDiffBack = 1.0f*(i - valleyLocs[lastValley]) / pulseWidth;
			valleyYDiffBack = (maxVal > minVal ? (inputs[i] - inputs[valleyLocs[lastValley]]) / (maxVal - minVal) : 0.0f);
		}
		else {
			valleyXDiffBack = 0;
			valleyYDiffBack = 0;
		}


		float valleyXDiffFor;
		float valleyYDiffFor;
		if (i < valleyLocs[valleyLocs.size() - 1]) {
			valleyXDiffFor = 1.0f*(valleyLocs[lastValley + 1] - i) / pulseWidth;
			valleyYDiffFor = (maxVal > minVal ? (inputs[valleyLocs[lastValley + 1]] - inputs[i]) / (maxVal - minVal) : 0.0f);
		}
		else {
			valleyXDiffFor = 0;
			valleyYDiffFor = 0;
		}

		float peakBackHypo = sqrt(peakXDiffBack*peakXDiffBack + peakYDiffBack*peakYDiffBack);
		float peakForHypo = sqrt(peakXDiffFor*peakXDiffFor + peakYDiffFor*peakYDiffFor);
		float valleyBackHypo = sqrt(valleyXDiffBack*valleyXDiffBack + valleyYDiffBack*valleyYDiffBack);
		float valleyForHypo = sqrt(valleyXDiffFor*valleyXDiffFor + valleyYDiffFor*valleyYDiffFor);

		if (CONV_PEAK_FEATURES_INCLUDE_WAVEFORM) {
			secondaryFeatures->push_back(inputs[i]);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_ALL_SLOPES) {
			if (peakBackHypo != 0)
				secondaryFeatures->push_back(peakYDiffBack / peakBackHypo);
			else
				secondaryFeatures->push_back(0.0f);

			if (peakForHypo != 0)
				secondaryFeatures->push_back(peakYDiffFor / peakForHypo);
			else
				secondaryFeatures->push_back(0.0f);

			if (valleyBackHypo != 0)
				secondaryFeatures->push_back(valleyYDiffBack / valleyBackHypo);
			else
				secondaryFeatures->push_back(0.0f);

			if (valleyForHypo != 0)
				secondaryFeatures->push_back(valleyYDiffFor / valleyForHypo);
			else
				secondaryFeatures->push_back(0.0f);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_NEAR_SLOPE) {
			if (i > peakLocs[0] && (i <= valleyLocs[0] || peakLocs[lastPeak] > valleyLocs[lastValley]) && peakBackHypo != 0)
				secondaryFeatures->push_back(peakYDiffBack / peakBackHypo);
			else if (i > valleyLocs[0] && (i <= peakLocs[0] || valleyLocs[lastValley] > peakLocs[lastPeak]) && valleyBackHypo != 0)
				secondaryFeatures->push_back(valleyYDiffBack / valleyBackHypo);
			else
				secondaryFeatures->push_back(0);
				
			if (i < peakLocs[peakLocs.size() - 1] && (i >= valleyLocs[valleyLocs.size() - 1] || peakLocs[lastPeak + 1] < valleyLocs[lastValley + 1]) && peakForHypo != 0)
				secondaryFeatures->push_back(peakYDiffFor / peakForHypo);
			else if (i < valleyLocs[valleyLocs.size() - 1] && (i >= peakLocs[peakLocs.size() - 1] || valleyLocs[lastValley + 1] < peakLocs[lastPeak + 1]) && valleyForHypo != 0)
				secondaryFeatures->push_back(valleyYDiffFor / valleyForHypo);
			else
				secondaryFeatures->push_back(0);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_FAR_SLOPE) {
			if (i > peakLocs[0] && i > valleyLocs[0] && (peakLocs[lastPeak] < valleyLocs[lastValley]) && peakBackHypo != 0)
				secondaryFeatures->push_back(peakYDiffBack / peakBackHypo);
			else if (i > peakLocs[0] && i > valleyLocs[0] && (valleyLocs[lastValley] < peakLocs[lastPeak]) && valleyBackHypo != 0)
				secondaryFeatures->push_back(valleyYDiffBack / valleyBackHypo);
			else
				secondaryFeatures->push_back(0);
				
			if (i < peakLocs[peakLocs.size() - 1] && i < valleyLocs[valleyLocs.size() - 1] && (peakLocs[lastPeak + 1] > valleyLocs[lastValley + 1]) && peakForHypo != 0)
				secondaryFeatures->push_back(peakYDiffFor / peakForHypo);
			else if (i < peakLocs[peakLocs.size() -1] && i < valleyLocs[valleyLocs.size() - 1] && (valleyLocs[lastValley + 1] > peakLocs[lastPeak + 1]) && valleyForHypo != 0)
				secondaryFeatures->push_back(valleyYDiffFor / valleyForHypo);
			else
				secondaryFeatures->push_back(0);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_X_POS) {
			//peakXBack
			if (i > peakLocs[0])
				secondaryFeatures->push_back(i - peakLocs[lastPeak]);
			else
				secondaryFeatures->push_back(0);

			//peakXFor
			if (i < peakLocs[peakLocs.size() - 1])
				secondaryFeatures->push_back(peakLocs[lastPeak + 1] - i);
			else
				secondaryFeatures->push_back(0);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_Y_POS) {
			//peakY
			if (i <= peakLocs[0])
				secondaryFeatures->push_back(inputs[peakLocs[0]] - inputs[i]);
			else if (i >= peakLocs[peakLocs.size() - 1])
				secondaryFeatures->push_back(inputs[peakLocs[peakLocs.size() - 1]] - inputs[i]);
			else
				secondaryFeatures->push_back((inputs[peakLocs[lastPeak]] + inputs[peakLocs[lastPeak + 1]]) / 2 - inputs[i]);

			//valleyY
			if (i <= valleyLocs[0])
				secondaryFeatures->push_back(inputs[valleyLocs[0]] - inputs[i]);
			else if (i >= valleyLocs[valleyLocs.size() - 1])
				secondaryFeatures->push_back(inputs[valleyLocs[valleyLocs.size() - 1]] - inputs[i]);
			else
				secondaryFeatures->push_back((inputs[valleyLocs[lastValley]] + inputs[valleyLocs[lastValley + 1]]) / 2 - inputs[i]);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_SLOPE_DIFF) {
			size_t backPoint = std::max((i > peakLocs[0] ? peakLocs[lastPeak] : 0), (i > valleyLocs[0] ? valleyLocs[lastValley] : 0));
			size_t forwardPoint = std::min((i <= peakLocs[peakLocs.size() - 1] ? (i > peakLocs[0] ? peakLocs[lastPeak + 1] : peakLocs[0]) : 9999), (i <= valleyLocs[valleyLocs.size() - 1] ? (i > valleyLocs[0] ? valleyLocs[lastValley + 1] : valleyLocs[0]) : 9999));
			if (backPoint == 0 || forwardPoint == 9999 || forwardPoint <= backPoint || inputs[forwardPoint] == inputs[backPoint]) {
				secondaryFeatures->push_back(0);
			}
			else {
				float slopeHeight = inputs[backPoint] + (inputs[forwardPoint] - inputs[backPoint]) * (i - backPoint) / (forwardPoint - backPoint);
				secondaryFeatures->push_back((inputs[i] - slopeHeight) / fabs(inputs[forwardPoint] - inputs[backPoint]));
			}
		}

		if (CONV_PEAK_FEATURES_INCLUDE_FIRST_DERIV) {
			secondaryFeatures->push_back(deriv1[i]);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_SECOND_DERIV) {
			secondaryFeatures->push_back(deriv2[i]);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_NORM_BY_CENTER) {
			if (inputs[centerPeak] > inputs[centerValley])
				secondaryFeatures->push_back(2.0f*(inputs[i] - inputs[centerValley]) / (inputs[centerPeak] - inputs[centerValley]) - 1.0f);
			else
				secondaryFeatures->push_back(0);
		}

		if (CONV_PEAK_FEATURES_INCLUDE_ALL_DERIV2_SLOPES) {
			float derivPeakYDiffBack;
			if (i > peakLocs[0]) {
				derivPeakYDiffBack = (maxDeriv2 > minDeriv2 ? (deriv2[i] - deriv2[peakLocs[lastPeak]]) / (maxDeriv2 - minDeriv2) : 0.0f);
			}
			else {
				derivPeakYDiffBack = 0;
			}

			float derivPeakYDiffFor;
			if (i < peakLocs[peakLocs.size() - 1]) {
				derivPeakYDiffFor = (maxDeriv2 > minDeriv2 ? (deriv2[peakLocs[lastPeak + 1]] - deriv2[i]) / (maxDeriv2 - minDeriv2) : 0.0f);
			}
			else {
				derivPeakYDiffFor = 0;
			}

			float derivValleyYDiffBack;
			if (i > valleyLocs[0]) {
				derivValleyYDiffBack = (maxDeriv2 > minDeriv2 ? (deriv2[i] - deriv2[valleyLocs[lastValley]]) / (maxDeriv2 - minDeriv2) : 0.0f);
			}
			else {
				derivValleyYDiffBack = 0;
			}


			float derivValleyYDiffFor;
			if (i < valleyLocs[valleyLocs.size() - 1]) {
				derivValleyYDiffFor = (maxDeriv2 > minDeriv2 ? (deriv2[valleyLocs[lastValley + 1]] - deriv2[i]) / (maxDeriv2 - minDeriv2) : 0.0f);
			}
			else {
				derivValleyYDiffFor = 0;
			}

			float derivPeakBackHypo = sqrt(peakXDiffBack*peakXDiffBack + derivPeakYDiffBack*derivPeakYDiffBack);
			float derivPeakForHypo = sqrt(peakXDiffFor*peakXDiffFor + derivPeakYDiffFor*derivPeakYDiffFor);
			float derivValleyBackHypo = sqrt(valleyXDiffBack*valleyXDiffBack + derivValleyYDiffBack*derivValleyYDiffBack);
			float derivValleyForHypo = sqrt(valleyXDiffFor*valleyXDiffFor + derivValleyYDiffFor*derivValleyYDiffFor);

			if (derivPeakBackHypo != 0)
				secondaryFeatures->push_back(derivPeakYDiffBack / derivPeakBackHypo);
			else
				secondaryFeatures->push_back(0.0f);

			if (derivPeakForHypo != 0)
				secondaryFeatures->push_back(derivPeakYDiffFor / derivPeakForHypo);
			else
				secondaryFeatures->push_back(0.0f);

			if (derivValleyBackHypo != 0)
				secondaryFeatures->push_back(derivValleyYDiffBack / derivValleyBackHypo);
			else
				secondaryFeatures->push_back(0.0f);

			if (derivValleyForHypo != 0)
				secondaryFeatures->push_back(derivValleyYDiffFor / derivValleyForHypo);
			else
				secondaryFeatures->push_back(0.0f);
		}
	}

	//local scaling
	for (size_t f = 0; f < numFeatures; f++) {
		if (!(*localScaleMask)[f])
			continue;
		float mean = 0;
		float stdev = 0;
		for (size_t i = 0; i < NUM_INPUTS; i++) {
			float val = (*secondaryFeatures)[f + i*numFeatures];
			mean += val;
			stdev += val*val;
		}
		mean /= NUM_INPUTS;
		stdev /= NUM_INPUTS;
		stdev = sqrt(stdev - mean*mean);
		for (size_t i = 0; i < NUM_INPUTS; i++) {
			(*secondaryFeatures)[f + i*numFeatures] = (stdev > 0 ? ((*secondaryFeatures)[f + i*numFeatures] - mean) / stdev : 0);
		}
	}

	return secondaryFeatures->size() / NUM_INPUTS;
}

size_t createSecondaryFixedFeatures(float* inputs, float* peaks, std::vector<float>* secondaryFeatures) {
	secondaryFeatures->clear();

	std::vector<size_t> peakLocs;
	std::vector<size_t> valleyLocs;
	float minVal = 9999;
	float maxVal = -9999;

	for (size_t i = 0; i < NUM_INPUTS; i++) {
		if (peaks[i] == 1)
			peakLocs.push_back(i);
		else if (peaks[i] == -1)
			valleyLocs.push_back(i);
		minVal = std::min(minVal, inputs[i]);
		maxVal = std::max(maxVal, inputs[i]);
	}

	//heartrate features
	float pulseWidthMean = 0;
	float pulseWidthStdev = 0;
	size_t numPulseWidths = 0;
	for (size_t i = 0; i + 1 < peakLocs.size(); i++) {
		float width = peakLocs[i + 1] - peakLocs[i];
		pulseWidthMean += width;
		pulseWidthStdev += width*width;
		numPulseWidths++;
	}
	for (size_t i = 0; i + 1 < valleyLocs.size(); i++) {
		float width = valleyLocs[i + 1] - valleyLocs[i];
		pulseWidthMean += width;
		pulseWidthStdev += width*width;
		numPulseWidths++;
	}
	pulseWidthMean /= numPulseWidths;
	pulseWidthStdev /= numPulseWidths;
	pulseWidthStdev = sqrt(pulseWidthStdev - pulseWidthMean*pulseWidthMean);

	secondaryFeatures->push_back(pulseWidthMean);
	secondaryFeatures->push_back(pulseWidthStdev);

	//slope features

	std::vector<size_t> slopeSizes = { 3, 6, 12, 24, 36 };
	
	for (size_t s = 0; s < slopeSizes.size(); s++) {
		//prePeak
		float mean = 0;
		float stdev = 0;
		float counts = 0;
		for (size_t i = 0; i < peakLocs.size(); i++) {
			if (peakLocs[i] < slopeSizes[s])
				continue;
			float slope = (inputs[peakLocs[i]] - inputs[peakLocs[i] - slopeSizes[s]]) / slopeSizes[s];
			mean += slope;
			stdev += slope*slope;
			counts++;
		}
		mean /= counts;
		stdev /= counts;
		stdev = (stdev > mean*mean ? sqrt(stdev - mean*mean) : 0);

		secondaryFeatures->push_back(mean);
		secondaryFeatures->push_back(stdev);

		//postPeak
		mean = 0;
		stdev = 0;
		counts = 0;
		for (size_t i = 0; i < peakLocs.size(); i++) {
			if (peakLocs[i] >= NUM_INPUTS - slopeSizes[s])
				continue;
			float slope = (inputs[peakLocs[i] + slopeSizes[s]] - inputs[peakLocs[i]]) / slopeSizes[s];
			mean += slope;
			stdev += slope*slope;
			counts++;
		}
		mean /= counts;
		stdev /= counts;
		stdev = (stdev > mean*mean ? sqrt(stdev - mean*mean) : 0);

		secondaryFeatures->push_back(mean);
		secondaryFeatures->push_back(stdev);

		//preValley
		mean = 0;
		stdev = 0;
		counts = 0;
		for (size_t i = 0; i < valleyLocs.size(); i++) {
			if (valleyLocs[i] < slopeSizes[s])
				continue;
			float slope = (inputs[valleyLocs[i]] - inputs[valleyLocs[i] - slopeSizes[s]]) / slopeSizes[s];
			mean += slope;
			stdev += slope*slope;
			counts++;
		}
		mean /= counts;
		stdev /= counts;
		stdev = (stdev > mean*mean ? sqrt(stdev - mean*mean) : 0);

		secondaryFeatures->push_back(mean);
		secondaryFeatures->push_back(stdev);

		//postValley
		mean = 0;
		stdev = 0;
		counts = 0;
		for (size_t i = 0; i < valleyLocs.size(); i++) {
			if (valleyLocs[i] >= NUM_INPUTS - slopeSizes[s])
				continue;
			float slope = (inputs[valleyLocs[i] + slopeSizes[s]] - inputs[valleyLocs[i]]) / slopeSizes[s];
			mean += slope;
			stdev += slope*slope;
			counts++;
		}
		mean /= counts;
		stdev /= counts;
		stdev = (stdev > mean*mean ? sqrt(stdev - mean*mean) : 0);

		secondaryFeatures->push_back(mean);
		secondaryFeatures->push_back(stdev);
	}

	return secondaryFeatures->size();
}

void scaleConvSecondaryFeatures(std::vector<FILE*> trainsets, std::vector<FILE*> valsets, std::vector<FILE*> testsets, size_t numFeatures, std::vector<bool>* globalScaleMask) {
#if !defined(USE_FIXED_FEATURES)
	size_t numInputs = NUM_INPUTS;
#else
	size_t numInputs = 1;
#endif
	for (size_t cv = 0; cv < trainsets.size(); cv++) {
		std::cout << "Scaling CV " << cv + 1 << ": ";
		std::vector<long double> featureMeans(numFeatures);
		std::vector<long double> featureStdevs(numFeatures);
		std::vector<float> features(numFeatures*numInputs);

		_fseeki64(trainsets[cv], 0, SEEK_SET);

		size_t numSamples = 0;
		fread(&numSamples, sizeof(size_t), 1, trainsets[cv]);
		size_t numOutputs = 0;
		if (USE_ALL_OUTPUTS)
			numOutputs = (SKIP_FIRST_OUTPUT ? NUM_OUTPUTS - 1 : NUM_OUTPUTS);
#ifdef PEAK_DATA_INCLUDED
		else if (NUM_SIDE_OUTPUT_PEAK_HEIGHTS > 0)
			numOutputs = NUM_SIDE_OUTPUT_PEAK_HEIGHTS * 2 + 1;
#endif
		else
			numOutputs = 1;
		size_t numPreFlags = numOutputs + 1;
		for (size_t s = 0; s < numSamples; s++) {
			_fseeki64(trainsets[cv], numPreFlags*sizeof(float), SEEK_CUR);
			fread(&features[0], sizeof(float), numFeatures*numInputs, trainsets[cv]);
			for (size_t i = 0; i < numInputs; i++) {
				for (size_t f = 0; f < numFeatures; f++) {
					long double val = (long double)features[f + i*numFeatures];
					/*
					if (val != val) {
						//TEST
						std::cout << "Invalid value on sample " << s + 1 << " input " << i + 1 << " feature " << f + 1 << std::endl;
						for (size_t v = 0; v < numFeatures*NUM_INPUTS; v++) {
							std::cout << features[v] << " ";
						}
						std::cout << std::endl;
						system("pause");
					}
					*/
					featureMeans[f] += val;
					featureStdevs[f] += val*val;
				}
			}
		}
		for (size_t f = 0; f < numFeatures; f++) {
			//std::cout << "F" << f + 1 << ": " << featureMeans[f] << " +\\- " << featureStdevs[f] << " ";	//TEST
			//well cast, idiot.
			featureMeans[f] /= (((long double)numSamples)*((long double)numInputs));
			featureStdevs[f] /= (((long double)numSamples)*((long double)numInputs));
			featureStdevs[f] = (featureStdevs[f] > featureMeans[f]*featureMeans[f] ? sqrt(featureStdevs[f] - featureMeans[f] * featureMeans[f]) : 0);
			std::cout << "F" << f + 1 << ": " << featureMeans[f] << " +\\- " << featureStdevs[f] << " ";
		}
		std::cout << std::endl;

		std::stringstream fss;
		fss << "featurenorms_" << cv + 1;
		std::ofstream featurefile(fss.str());
		for (size_t f = 0; f < numFeatures; f++) {
			featurefile << featureMeans[f] << " ";
		}
		featurefile << std::endl;
		for (size_t f = 0; f < numFeatures; f++) {
			featurefile << featureStdevs[f] << " ";
		}
		featurefile << std::endl;
		featurefile.close();

		for (size_t type = 0; type < 3; type++) {
			FILE* typeFile;
			if (type == 0)
				typeFile = trainsets[cv];
			else if (type == 1)
				typeFile = valsets[cv];
			else
				typeFile = testsets[cv];

			_fseeki64(typeFile, 0, SEEK_SET);
			fread(&numSamples, sizeof(size_t), 1, typeFile);
			for (size_t s = 0; s < numSamples; s++) {
				_fseeki64(typeFile, numPreFlags*sizeof(float), SEEK_CUR);
				__int64 initPos = _ftelli64(typeFile);
				fread(&features[0], sizeof(float), numFeatures*numInputs, typeFile);
				_fseeki64(typeFile, initPos, SEEK_SET);
				for (size_t i = 0; i < numInputs; i++) {
					for (size_t f = 0; f < numFeatures; f++) {
						float feat = features[f + i*numFeatures];
#if !defined(USE_FIXED_FEATURES)
						if (globalScaleMask->size() <= f || (*globalScaleMask)[f]) {
							if (featureStdevs[f] > 0)
								feat = (feat - (float)featureMeans[f]) / ((float)featureStdevs[f]);
							else
								feat = 0;
						}
#else
						feat = (feat - (float)featureMeans[f]) / ((float)featureStdevs[f]);
#endif
						fwrite(&feat, sizeof(float), 1, typeFile);
					}
				}
			}
		}
	}
}

#ifdef PEAK_DATA_INCLUDED
void calculatePeakOutputs(float* inputs, float* peaks, std::vector<float>* outputs) {
	std::vector<size_t> peakLocs;
	size_t centerPeak = 0;
	int minDistFromCenter = 9999;
	for (size_t i = 0; i < NUM_INPUTS; i++) {
		if (peaks[i] == 1)
			peakLocs.push_back(i);
		if (abs((int)i - NUM_INPUTS / 2) < minDistFromCenter) {
			centerPeak = peakLocs.size() - 1;
			minDistFromCenter = abs((int)i - NUM_INPUTS / 2);
		}
	}

	outputs->clear();

	for (int i = -NUM_SIDE_OUTPUT_PEAK_HEIGHTS; i <= NUM_SIDE_OUTPUT_PEAK_HEIGHTS; i++) {
		int peakPos = (int)centerPeak + i;
		if (peakPos < 0)
			outputs->push_back(inputs[peakLocs[0]]);
		else if (peakPos > peakLocs.size() - 1)
			outputs->push_back(inputs[peakLocs[peakLocs.size() - 1]]);
		else
			outputs->push_back(inputs[peakLocs[peakPos]]);
	}
}
#endif
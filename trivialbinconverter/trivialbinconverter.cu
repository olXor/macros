#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <random>
#include "trivialbinconverterPeakFinder.cuh"

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

#define HEADER_SIZE 8
#define NUM_INPUTS_1 512	//includes peak information, if present
#define NUM_SHIFT_INPUTS_1 512	//does NOT include peak information, if present
#define NUM_OUTPUTS_1 2
#define NUM_INPUTS_2 512
std::vector<size_t> outputIndices = { 0, 1 };
#define NUM_EXTRA_OUTPUTS 0

#define MAX_INPUT_SHIFT 0

#define INTERPOLATE_INPUT_SCALING (116.0f/70.0f)
#define NUM_SCALE_SHIFT_INPUTS_1 ((size_t)(NUM_SHIFT_INPUTS_1*INTERPOLATE_INPUT_SCALING))

#define INCLUDE_INVERTED_WAVEFORM 0
#define INCLUDE_CALCULATED_PEAKS 0

int main() {
	srand((size_t)time(NULL));
	std::string outfname;
	std::cout << "Enter output file prefix: ";
	std::cin >> outfname;

	std::string filelist = "filelist";
	std::ifstream infilelist(datastring + filelist);
	if (!infilelist.is_open()) {
		std::cout << "Couldn't open file list " << datastring << filelist << std::endl;
	}

	std::string fname;
	std::vector<float> columns(NUM_INPUTS_1 + NUM_OUTPUTS_1);
	std::vector<float> unscaledWaveform;
	std::vector<float> waveform;
	std::vector<float> unscaledPeaks;
	std::vector<float> peaks;
	while (std::getline(infilelist, fname)) {
		std::cout << "Converting " << fname << std::endl;
		FILE* infile = fopen((datastring + fname).c_str(), "rb");
		std::stringstream outss;
		std::string localName;
		if (fname.find_last_of("\\/") != 0)
			localName = fname.substr(fname.find_last_of("\\/") + 1, std::string::npos);
		else
			localName = fname;
		outss << datastring << outfname << "_" << localName;
		FILE* outfile = fopen(outss.str().c_str(), "wb");
		char* header[HEADER_SIZE];
		fread(&header, HEADER_SIZE, 1, infile);
		fwrite(&header, HEADER_SIZE, 1, outfile);

		while (fread(&columns[0], sizeof(float), NUM_OUTPUTS_1 + NUM_INPUTS_1, infile) == NUM_OUTPUTS_1 + NUM_INPUTS_1) {
			for (size_t inv = 0; inv < (INCLUDE_INVERTED_WAVEFORM ? 2 : 1); inv++) {
				size_t inputShift = 0;
				if (NUM_INPUTS_2 + 2*MAX_INPUT_SHIFT > NUM_SCALE_SHIFT_INPUTS_1) {
					std::cout << "NUM_INPUTS_2 + 2*MAX_INPUT_SHIFT > NUM_SCALE_SHIFT_INPUTS_1" << std::endl;
					system("pause");
					return 0;
				}
				if (MAX_INPUT_SHIFT > 0) {
					inputShift = NUM_SCALE_SHIFT_INPUTS_1 / 2 - NUM_INPUTS_2 / 2 + (rand() % 2 * MAX_INPUT_SHIFT) - MAX_INPUT_SHIFT;
					if (inputShift + NUM_INPUTS_2 > NUM_SCALE_SHIFT_INPUTS_1) {
						std::cout << "inputShift + NUM_INPUTS_2 > NUM_SCALE_SHIFT_INPUTS_1; inputShift: " << inputShift << std::endl;
						system("pause");
						return 0;
					}
				}
				else
					inputShift = NUM_SCALE_SHIFT_INPUTS_1 / 2 - NUM_INPUTS_2 / 2;
				for (size_t o = 0; o < outputIndices.size(); o++) {
					fwrite(&columns[outputIndices[o]], sizeof(float), 1, outfile);
				}
				float dum = 0;
				for (size_t o = 0; o < NUM_EXTRA_OUTPUTS; o++) {
					fwrite(&dum, sizeof(float), 1, outfile);
				}
				unscaledWaveform.clear();
				unscaledWaveform.resize(NUM_SHIFT_INPUTS_1);
				for (size_t w = 0; w < unscaledWaveform.size(); w++) {
					unscaledWaveform[w] = columns[NUM_OUTPUTS_1 + w];
				}
				waveform.clear();
				waveform.resize(NUM_SCALE_SHIFT_INPUTS_1);
				interpolate(&unscaledWaveform, &waveform, waveform.size());

				fwrite(&waveform[inputShift], sizeof(float), NUM_INPUTS_2, outfile);

				if (INCLUDE_CALCULATED_PEAKS) {
					peaks.clear();
					peaks.resize(waveform.size());
					findPeaksAndValleys(&waveform, &peaks);

					fwrite(&peaks[inputShift], sizeof(float), NUM_INPUTS_2, outfile);
				}

				if (inv < (INCLUDE_INVERTED_WAVEFORM ? 2 : 1) - 1) {
					for (size_t i = 0; i < NUM_INPUTS_1; i++) {
						columns[NUM_OUTPUTS_1 + i] = -columns[NUM_OUTPUTS_1 + i];
					}
				}
			}
		}

		fclose(infile);
		fclose(outfile);
	}

	system("pause");
}

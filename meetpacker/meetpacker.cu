
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/meetpacker/"

#define HEADER_SIZE 4 

#define NUM_INPUTS 512
#define NUM_OUTPUTS 16
#define NUM_PEAKLOCS 30	//same as valleys
#define NUM_FEATURES 591

void meetPackerConvertFiles(FILE* oldfile, FILE* newfile);

int main() {
	std::string filelist = "meetfilelist";
	std::ifstream meetfile(datastring + filelist);
	if (!meetfile.is_open()) {
		std::cout << "Couldn't open file list " << datastring << filelist << std::endl;
		system("pause");
		return;
	}

	std::string meetline;
	while (std::getline(meetfile, meetline)) {
		std::stringstream lss(meetline);
		std::string oldfname, newfname;
		lss >> oldfname >> newfname;

		FILE* oldfile = fopen((datastring + oldfname).c_str(), "rb");
		FILE* newfile = fopen((datastring + newfname).c_str(), "wb");

		meetPackerConvertFiles(oldfile, newfile);
		fclose(oldfile);
		fclose(newfile);
	}
}

void meetPackerConvertFiles(FILE* oldfile, FILE* newfile) {
	fseek(oldfile, HEADER_SIZE, SEEK_SET);
	fseek(newfile, 0, SEEK_SET);

	size_t dum = 0;
	fwrite(&dum, sizeof(size_t), 1, newfile);
	fwrite(&dum, sizeof(size_t), 1, newfile);	//dummy header

	int numColumns;
	fread(&numColumns, sizeof(int), 1, oldfile);

	if (numColumns != NUM_OUTPUTS + NUM_INPUTS + 2 * NUM_PEAKLOCS + NUM_FEATURES + 1) {
		std::cout << "Invalid number of columns in at least one file: expected " << NUM_OUTPUTS + NUM_INPUTS + 2 * NUM_PEAKLOCS + NUM_FEATURES + 1 << " got " << numColumns << std::endl;
		system("pause");
		return;
	}

	std::vector<float> columns(numColumns);

	while (fread(&columns[0], sizeof(float), numColumns, oldfile) == numColumns) {
		fwrite(&columns[0], sizeof(float), 1 + NUM_OUTPUTS + NUM_INPUTS, newfile);
		std::vector<float> peaks(NUM_INPUTS);
		for (size_t i = 0; i < NUM_PEAKLOCS; i++) {
			float peakLoc = columns[1 + NUM_OUTPUTS + NUM_INPUTS + i];
			if (peakLoc == peakLoc)
				peaks[(size_t)peakLoc] = 1;
			float valleyLoc = columns[1 + NUM_OUTPUTS + NUM_INPUTS + NUM_PEAKLOCS + i];
			if (valleyLoc == valleyLoc)
				peaks[(size_t)valleyLoc] = -1;
		}

		fwrite(&peaks[0], sizeof(float), NUM_INPUTS, newfile);
	}
}
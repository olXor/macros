#include "hip/hip_runtime.h"
#include "structdef.cuh"
#define _USE_MATH_DEFINES
#include <math.h>

#define datastring "D:/stopSearch/data/"
//#define signalstring "eventSaveBTag3JetsignalWB_350-200_1871520"
//#define backgroundstring "eventSaveBTag3Jetbackground_7625071"
//#define signalstring "convFixPhiBTag_WBSignal_350-200_1994132"
//#define signalstring "convFixPhiBTag_signalTN_185-5_1300000"
//#define backgroundstring "convFixPhiBTag_background_7625071"
//#define signalstring "../weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/nhistL12N0_g10_type0"
//#define backgroundstring "../weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/nhistL12N0_g10_type1"
//#define signalstring "../weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/nhistL12N0_l10g-10_type0"
//#define backgroundstring "../weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/nhistL12N0_l10g-10_type1"
//#define signalstring "../weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/nhistL12N0_l-10_type0"
//#define backgroundstring "../weights/deep_350-200_FixPhiWBMatchedNoMETEtaBTag_L1_2e-5/nhistL12N0_l-10_type1"
//#define signalstring "convRelPhiBTag_TNSignal_185-5_1300000"
//#define backgroundstring "convRelPhiBTag_background_7625071"
//#define signalstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL4N11_g100_type0"
//#define backgroundstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL4N11_g100_type1"
//#define signalstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL12N0_l-0p4_type0"
//#define backgroundstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL12N0_l-0p4_type1"
//#define signalstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL12N0_type0"
//#define backgroundstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL12N0_type1"
//#define signalstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL12N0_l-0p4_all"
//#define backgroundstring "../weights/deep_185-5_FixPhiTNNoMETEtaBTag_L1_2e-5_dropout/nhistL12N0_all"
#define signalstring "../weights/deep_750-1_FixPhiTNBTag_L1_2e-5/nhistL12N0_l-20_type0"
#define backgroundstring "../weights/deep_750-1_FixPhiTNBTag_L1_2e-5/nhistL12N0_g-20_type1"

#define NORMALIZE_SBRATIO 0

#define WEIGHT_1 (5*0.00184*35900/1968700)
//#define WEIGHT_1 (5*0.0834*35900/1968700)
//#define WEIGHT_1 (5*2.38*35900/1300000)
#define WEIGHT_2 (5*24.6*35900/7329772)

#define CUSTOM_VAR_SIZE 2

/*
#define datastring "D:/stopSearch/data/"
#define signalstring "nhistL5N10_type0"
#define backgroundstring "nhistL5N10_type1"
*/

#define NUM_BINS 20

bool acceptEvent(std::vector<float>* vals) {
	//remember to add one because first val is network output
	size_t preVars = 1;	//number of variables coming before input (ie. network output)
	float metPt = exp((*vals)[0 + preVars]);
	float metPhi = 0;
	float l1Pt = exp((*vals)[1 + preVars]);
	float l2Pt = exp((*vals)[3 + preVars]);
	float j1Pt = exp((*vals)[5 + preVars]);
	float j2Pt = exp((*vals)[8 + preVars]);

	//FixPhiBTag
	float l1Phi = (*vals)[11 + preVars]*(*vals)[12 + preVars];
	float l2Phi = (*vals)[13 + preVars]*(*vals)[14 + preVars];
	float j1Phi = (*vals)[15 + preVars]*(*vals)[16 + preVars];
	float j2Phi = (*vals)[17 + preVars]*(*vals)[18 + preVars];

	//RelPhiBTag
	/*
	float l1Phi = (*vals)[11 + preVars];
	float l2Phi = (*vals)[12 + preVars]*(*vals)[13 + preVars];
	float j1Phi = (*vals)[14 + preVars]*(*vals)[15 + preVars];
	float j2Phi = (*vals)[16 + preVars]*(*vals)[17 + preVars];
	*/

	float l1Eta = (*vals)[2 + preVars];
	float l2Eta = (*vals)[4 + preVars];

	//return std::cos(l1Phi) < -0.5 && std::cos(l2Phi) < -0.5 && metPt + 2 * l2Pt > 160;
	//return fabs(l1Phi - l2Phi) < M_PI/4 && fabs(l1Phi) < M_PI/2 && fabs(l2Phi) < M_PI/2;
	return true;
}

std::vector<float> getCustomVars(std::vector<float>* vals) {
	std::vector<float> vars(CUSTOM_VAR_SIZE);
	//remember to add one because first val is network output
	size_t preVars = 1;	//number of variables coming before input (ie. network output)
	float metPt = exp((*vals)[0 + preVars]);
	float metPhi = 0;
	float l1Pt = exp((*vals)[1 + preVars]);
	float l2Pt = exp((*vals)[3 + preVars]);
	float j1Pt = exp((*vals)[5 + preVars]);
	float j2Pt = exp((*vals)[8 + preVars]);

	//FixPhiBTag
	float l1Phi = (*vals)[11 + preVars]*(*vals)[12 + preVars];
	float l2Phi = (*vals)[13 + preVars]*(*vals)[14 + preVars];
	float j1Phi = (*vals)[15 + preVars]*(*vals)[16 + preVars];
	float j2Phi = (*vals)[17 + preVars]*(*vals)[18 + preVars];

	//RelPhiBTag
	/*
	float l1Phi = (*vals)[11 + preVars];
	float l2Phi = (*vals)[12 + preVars]*(*vals)[13 + preVars];
	float j1Phi = (*vals)[14 + preVars]*(*vals)[15 + preVars];
	float j2Phi = (*vals)[16 + preVars]*(*vals)[17 + preVars];
	*/

	float l1Eta = (*vals)[2 + preVars];
	float l2Eta = (*vals)[4 + preVars];

	float metPx = metPt*cos(metPhi);
	float metPy = metPt*sin(metPhi);
	float l1Px = l1Pt*cos(l1Phi);
	float l1Py = l1Pt*sin(l1Phi);
	float l2Px = l2Pt*cos(l2Phi);
	float l2Py = l2Pt*sin(l2Phi);
	float j1Px = j1Pt*cos(j1Phi);
	float j1Py = j1Pt*sin(j1Phi);
	float j2Px = j2Pt*cos(j2Phi);
	float j2Py = j2Pt*sin(j2Phi);

	float recoilPx = metPx + l1Px + l2Px + j1Px + j2Px;
	float recoilPy = metPy + l1Py + l2Py + j1Py + j2Py;
	float recoilPt = sqrt(recoilPx*recoilPx + recoilPy*recoilPy);

	float phiSum = fabs(l1Phi) + fabs(l2Phi);

	float ISRPt = 0;
	if ((*vals)[8] == 1 && (*vals)[11] == 1)
		ISRPt = recoilPt;
	else if ((*vals)[8] == -1)
		ISRPt = j1Pt;
	else if ((*vals)[11] == -1)
		ISRPt = j2Pt;

	float lepPhiDiff = fabs(l1Phi - l2Phi);
	if (lepPhiDiff > M_PI)
		lepPhiDiff = 2 * M_PI - lepPhiDiff;
	float lepPhiSum = fabs(l1Phi) + fabs(l2Phi);
	float lepEtaDiff = fabs(l1Eta - l2Eta);
	//vars[0] = lepPhiDiff;// fabs(l1Eta - l2Eta);
	//vars[0] = l1Phi*l2Phi;
	//vars[0] = 3 * std::log(metPt) + std::log(l1Pt) + std::log(l2Pt);
	//vars[0] = l1Eta - l2Eta;
	vars[0] = std::log(metPt);
	vars[1] = lepPhiSum;

	return vars;
}

void convertInputs(std::vector<float>* inputs) {
	return;
	(*inputs)[0] = std::log((*inputs)[0]);
	(*inputs)[3] = std::log((*inputs)[3]);
	(*inputs)[6] = std::log((*inputs)[6]);
	(*inputs)[9] = std::log((*inputs)[9]);
	(*inputs)[13] = std::log((*inputs)[13]);
	(*inputs)[17] = std::log((*inputs)[17]);

	//non-overlapping phi combinations
	bool addBTags = true;
	for (size_t i = 1; i < 6; i++) {
		size_t numExtraVars = (i > 3 && addBTags ? i - 3 : 0);	//b-tags
		float firstPhi = (*inputs)[1];	//MET
		float secondPhi = (*inputs)[3 * i + 1 + numExtraVars];
		float phiDiff = secondPhi - firstPhi;
		float convPhi = fabs(firstPhi - secondPhi);
		if (phiDiff > M_PI)
			phiDiff -= 2 * M_PI;
		if (phiDiff < -M_PI)
			phiDiff += 2 * M_PI;
		float phiSign = (phiDiff > 0 ? 1 : -1);
		if (convPhi > M_PI)
			convPhi = 2 * M_PI - convPhi;
		if (firstPhi == 0 || secondPhi == 0)
			convPhi = 0;
		(*inputs)[3 * i + 1 + numExtraVars] = convPhi;
	}
}

int main() {
	std::string suffix;
	std::cout << "Enter suffix (\".\") for none: ";
	std::cin >> suffix;
	std::stringstream sigss;
	sigss << datastring << signalstring << (suffix != "." ? suffix : "");
	std::ifstream sigstream(sigss.str());
	std::stringstream backss;
	backss << datastring << backgroundstring << (suffix != "." ? suffix : "");
	std::ifstream backstream(backss.str());
	std::stringstream outss;
	outss << datastring << "plotdist_save";
	std::ofstream outstream(outss.str());
	std::stringstream outss2;
	outss2 << datastring << "plotdist_save2";
	std::ofstream outstream2(outss2.str());
	std::stringstream outss3;
	outss3 << datastring << "plotdist_save3";
	std::ofstream outstream3(outss3.str());
	std::stringstream outss4;
	outss4 << datastring << "plotdist_save4";
	std::ofstream outstream4(outss4.str());

	std::vector<size_t> vars;
	bool doneGetVars = false;
	bool manualBounds = false;
	std::cout << "Enter histogram boundaries manually? ";
	std::cin >> manualBounds;
	std::vector<float> manualMins;
	std::vector<float> manualMaxes;
	bool useCustomVars = false;
	std::cout << "Use hardcoded custom variables? ";
	std::cin >> useCustomVars;
	if (!useCustomVars) {
		while (!doneGetVars) {
			std::cout << "Enter variable to plot: ";
			size_t var;
			std::cin >> var;
			vars.push_back(var);
			if (manualBounds) {
				std::cout << "Enter min: ";
				float min;
				std::cin >> min;
				manualMins.push_back(min);
				std::cout << "Enter max: ";
				float max;
				std::cin >> max;
				manualMaxes.push_back(max);
			}
			std::cout << "Done entering variables? ";
			std::cin >> doneGetVars;
		}
	}
	else {
		for (size_t i = 0; i < CUSTOM_VAR_SIZE; i++) {
			vars.push_back(0);
			if (manualBounds) {
				std::cout << "Enter min " << i + 1 << ": ";
				float min;
				std::cin >> min;
				manualMins.push_back(min);
				std::cout << "Enter max " << i + 1 << ": ";
				float max;
				std::cin >> max;
				manualMaxes.push_back(max);
			}
		}
	}

	std::vector<std::ifstream*> streams;
	streams.push_back(&sigstream);
	streams.push_back(&backstream);

	std::vector<std::vector<std::vector<float>>> vals(streams.size());
	std::string line;
	std::vector<float> curVals;
	std::vector<float> minVals(vars.size());
	std::vector<float> maxVals(vars.size());
	std::vector<size_t> numBins(vars.size());
	for (size_t i = 0; i < vars.size(); i++) {
		minVals[i] = 99999;
		maxVals[i] = -99999;
		numBins[i] = NUM_BINS;
	}

	for (size_t i = 0; i < streams.size(); i++) {
		while (std::getline((*streams[i]), line)) {
			curVals.clear();
			std::stringstream lss(line);
			float val;
			while (lss >> val)
				curVals.push_back(val);
			convertInputs(&curVals);
			std::vector<float> varVals;
			if (!useCustomVars) {
				for (size_t v = 0; v < vars.size(); v++)
					varVals.push_back(curVals[vars[v]]);
			}
			else
				varVals = getCustomVars(&curVals);
			if (acceptEvent(&curVals))
				vals[i].push_back(varVals);

			for (size_t j = 0; j < vars.size(); j++) {
				minVals[j] = std::min(minVals[j], varVals[j]);
				maxVals[j] = std::max(maxVals[j], varVals[j]);
			}
		}
	}

	std::vector<SparseHistogram> hists(streams.size());
	if (manualBounds) {
		minVals = manualMins;
		maxVals = manualMaxes;
	}
	for (size_t i = 0; i < hists.size(); i++) {
		hists[i].initHistogram(minVals, maxVals, numBins);
	}

	for (size_t i = 0; i < vals.size(); i++) {
		for (size_t j = 0; j < vals[i].size(); j++) {
			float weight = (i == 0 ? WEIGHT_1 : WEIGHT_2);
			hists[i].fill(vals[i][j], weight);
		}
	}

	if (vars.size() == 1) {
		for (size_t i = 0; i < NUM_BINS; i++) {
			float posVal = i*(hists[0].maxes[0] - hists[0].mins[0]) / hists[0].numBins[0] + hists[0].mins[0] + (hists[0].maxes[0] - hists[0].mins[0]) / hists[0].numBins[0] / 2;
			std::vector<size_t> pos;
			pos.push_back(i);
			outstream << posVal << " " << hists[0].getWeightOfBin(&pos) / hists[0].totalWeight << " " << hists[1].getWeightOfBin(&pos) / hists[1].totalWeight << " " << (hists[0].getWeightOfBin(&pos))/(hists[1].getWeightOfBin(&pos)) << std::endl;
		}
	}
	else if (vars.size() == 2) {
		outstream << "# " << minVals[0] << " " << maxVals[0] << " " << minVals[1] << " " << maxVals[1] << " " << numBins[0] << " " << numBins[1] << std::endl;
		for (size_t i = 0; i < numBins[0]; i++) {
			for (size_t j = 0; j < numBins[1]; j++) {
				std::vector<size_t> pos = { i, j };
				float normRatio = (NORMALIZE_SBRATIO ? hists[1].totalWeight / hists[0].totalWeight : 1);
				outstream << normRatio*hists[0].getWeightOfBin(&pos) / hists[1].getWeightOfBin(&pos) << " ";
			}
			outstream << std::endl;
		}

		outstream2 << "# " << minVals[0] << " " << maxVals[0] << " " << minVals[1] << " " << maxVals[1] << " " << numBins[0] << " " << numBins[1] << std::endl;
		for (size_t i = 0; i < numBins[0]; i++) {
			for (size_t j = 0; j < numBins[1]; j++) {
				std::vector<size_t> pos = { i, j };
				outstream2 << hists[0].getWeightOfBin(&pos) << " ";
			}
			outstream2 << std::endl;
		}

		outstream3 << "# " << minVals[0] << " " << maxVals[0] << " " << minVals[1] << " " << maxVals[1] << " " << numBins[0] << " " << numBins[1] << std::endl;
		for (size_t i = 0; i < numBins[0]; i++) {
			for (size_t j = 0; j < numBins[1]; j++) {
				std::vector<size_t> pos = { i, j };
				outstream3 << hists[1].getWeightOfBin(&pos) << " ";
			}
			outstream3 << std::endl;
		}

		outstream4 << "# " << minVals[0] << " " << maxVals[0] << " " << minVals[1] << " " << maxVals[1] << " " << numBins[0] << " " << numBins[1] << std::endl;
		for (size_t i = 0; i < numBins[0]; i++) {
			for (size_t j = 0; j < numBins[1]; j++) {
				std::vector<size_t> pos = { i, j };
				outstream4 << hists[0].getWeightOfBin(&pos) + hists[1].getWeightOfBin(&pos) << " ";
			}
			outstream4 << std::endl;
		}

		std::cout << "Total weights: Signal: " << hists[0].totalWeight << " Background: " << hists[1].totalWeight << std::endl;
	}
	else
		throwError("Too many variables");
}
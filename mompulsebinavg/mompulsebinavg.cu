
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <cstdio>
#include <list>

//#define datastring "D:/momMIMIC/rawdata/"
#define datastring "rawdata/"

struct AggregateInfo {
	size_t windowSize;
	size_t numWindows;	//number of windows to average
	size_t headerSize;	//in bytes
	size_t numPreVariables;
};

void writeAggregateInterval(FILE* infile, FILE* outfile, AggregateInfo info);

//converts a series of .bin files to .bin files with averaged pulses (and stdevs)
int main() {
	std::string trainfname;
	std::cout << "Enter trainset name: ";
	std::cin >> trainfname;
	trainfname = datastring + trainfname;
	
	std::string outprefix;
	std::cout << "Enter output prefix: ";
	std::cin >> outprefix;

	AggregateInfo info;

	std::cout << "Enter window size: ";
	std::cin >> info.windowSize;

	std::cout << "Enter number of windows to average: ";
	std::cin >> info.numWindows;

	std::cout << "Enter number of additional input/output variables per window: ";
	std::cin >> info.numPreVariables;

	std::cout << "Enter size of header (in bytes): ";
	std::cin >> info.headerSize;

	std::ifstream trainfile(trainfname);
	std::string line;
	while (std::getline(trainfile, line)) {
		std::stringstream lss(line);
		std::string fname;
		lss >> fname;
		std::cout << "Processing file " << fname << std::endl;
		FILE* binfile = fopen((datastring + fname).c_str(), "rb");
		FILE* outbinfile = fopen((datastring + outprefix + fname).c_str(), "wb");

		writeAggregateInterval(binfile, outbinfile, info);
		fclose(binfile);
		fclose(outbinfile);
	}

	system("pause");
}

void writeAggregateInterval(FILE* infile, FILE* outfile, AggregateInfo info) {
	fseek(infile, info.headerSize, SEEK_SET);

	size_t numColumns = info.numPreVariables + info.windowSize;
	std::vector<float> columns(numColumns);
	std::vector<std::vector<float>> aggColumns(info.numWindows);
	size_t curWindow = 0;
	
	std::vector<float> windowAverages(info.windowSize);
	std::vector<float> windowStdevs(info.windowSize);

	while (fread(&columns[0], sizeof(float), columns.size(), infile) == columns.size()) {
		aggColumns[curWindow] = columns;
		curWindow++;
		
		if (curWindow >= aggColumns.size()) {
			for (size_t i = 0; i < info.numPreVariables; i++) {
				float avg = 0;
				for (size_t j = 0; j < info.numWindows; j++) {
					avg += aggColumns[j][i];
				}
				avg /= info.numWindows;
				fwrite(&avg, sizeof(float), 1, outfile);
			}

			float maxAvg = -9999;
			float minAvg = 9999;
			for (size_t i = 0; i < info.windowSize; i++) {
				float avg = 0;
				float stdev = 0;
				for (size_t j = 0; j < info.numWindows; j++) {
					float val = aggColumns[j][i + info.numPreVariables];
					avg += val;
					stdev += val*val;
				}
				avg /= info.numWindows;
				stdev /= info.numWindows;
				stdev -= avg*avg;

				stdev = sqrt((stdev > 0 ? stdev : 0));

				windowAverages[i] = avg;
				windowStdevs[i] = stdev;
				maxAvg = std::max(avg, maxAvg);
				minAvg = std::min(avg, minAvg);
			}
			for (size_t i = 0; i < info.windowSize; i++) {
				if (maxAvg > minAvg) {
					windowAverages[i] = 2.0f * (windowAverages[i] - minAvg) / (maxAvg - minAvg) - 1.0f;
					windowStdevs[i] /= ((maxAvg - minAvg) / 2.0f);
				}
				else {
					windowAverages[i] = 0;
					windowStdevs[i] = 0;
				}
				fwrite(&windowAverages[i], sizeof(float), 1, outfile);
				fwrite(&windowStdevs[i], sizeof(float), 1, outfile);
			}

			curWindow = 0;
		}
	}
}


#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <list>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

#define COMMA_DELIMITER

#define DATA_EPOCHTIME_DIVISOR 1000
#define INTERVAL_SIZE 6900

int main() {
	std::string datafname;
	std::cout << "Enter data file: ";
	std::cin >> datafname;

	std::string dataoutfname;
	std::cout << "Enter new data file: ";
	std::cin >> dataoutfname;

	std::ifstream datafile(datastring + datafname);
	if (!datafile.is_open()) {
		std::cout << "Can't find datafile" << std::endl;
		system("pause");
		return 0;
	}

	std::ofstream outfile(datastring + dataoutfname);

	std::string line;
	while (std::getline(datafile, line)) {
		long long epochTime;
		std::string fname;
		float output;
		(std::stringstream(line)) >> epochTime >> fname >> output;
		std::cout << "Starting: " << epochTime << " " << fname << " " << output << std::endl;

		std::ifstream infile(datastring + fname);
		std::string inLine;
		bool preData = false;
		size_t lineNum = 1;
		bool foundInterval = false;
		size_t intervalStart = 0;
		size_t intervalEnd = 0;
		std::getline(infile, inLine);	//header
		while (std::getline(infile, inLine)) {
			std::string tok;
			lineNum++;
#ifdef COMMA_DELIMITER
			std::getline((std::stringstream(inLine)), tok, ',');
#else
			(std::stringstream(inLine)) >> tok;
#endif
			long long lineEpochTime;
			(std::stringstream(tok)) >> lineEpochTime;
			lineEpochTime /= DATA_EPOCHTIME_DIVISOR;

			if (lineEpochTime < epochTime && !preData)
				preData = true;
			else if (((lineEpochTime > epochTime && preData) || (lineEpochTime == epochTime)) && !foundInterval) {
				foundInterval = true;
				intervalStart = lineNum;
				intervalEnd = lineNum;
			}
			else if (foundInterval) {
				intervalEnd++;
				if (intervalEnd >= intervalStart + INTERVAL_SIZE)
					break;
			}
			else if (lineEpochTime > epochTime && !preData) {
				break;
			}
		}
		if (foundInterval) {
			outfile << fname << " " << intervalStart << " " << intervalEnd << " " << output << std::endl;
			std::cout << fname << " " << intervalStart << " " << intervalEnd << " " << output << std::endl;
		}
		else if (!preData)
			std::cout << "File starts after interval" << std::endl;
		else
			std::cout << "File ends before interval" << std::endl;
	}

	system("pause");
}

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>

#define SENSOR_LENGTH 336
//#define datastring "D:/momRegression/rawdata/"
#define datastring ""

void saveOutputFile1(std::string infname, std::ofstream* outfile);
void saveOutputFile2(std::string infname, std::ofstream* outfile);
void saveOutputFile3(std::string infname, std::ofstream* outfile);
void saveOutputFile4(std::string infname, FILE* outbinfile);
void saveOutputFile5(std::string infname, FILE* outbinfile);
void saveOutputFile7(std::string infname, FILE* outbinfile);
void saveOutputFile8(std::string infname, FILE* outbinfile);

int main() {
	std::string trainfname;
	std::cout << "Enter trainset name: ";
	std::cin >> trainfname;
	trainfname = datastring + trainfname;
	
	std::string outfname;
	std::cout << "Enter name of output file: ";
	std::cin >> outfname;
	std::string shortOutFname = outfname;
	outfname = datastring + outfname;

	size_t inputType = 0;
	std::cout << "Enter input type: ";
	std::cin >> inputType;

	std::ofstream intervalfile(outfname + "_interval");

	std::ifstream infile(trainfname);
	std::string line;
	size_t num = 0;
	while (std::getline(infile, line)) {
		std::cout << "Converting file: " << line << std::endl;
		num++;
		std::stringstream lss(line);
		std::string fname;
		lss >> fname;
		fname = datastring + fname;
		std::stringstream outfss;
		outfss << datastring << shortOutFname << "_" << num;
		if (inputType == 1) {
			std::ofstream outfile(outfss.str());
			saveOutputFile1(fname, &outfile);
		}
		else if (inputType == 2) {
			std::ofstream outfile(outfss.str());
			saveOutputFile2(fname, &outfile);
		}
		else if (inputType == 3) {
			std::ofstream outfile(outfss.str());
			saveOutputFile3(fname, &outfile);
		}
		else if (inputType == 4) {
			FILE* outfile = fopen(outfss.str().c_str(), "wb");
			saveOutputFile4(fname, outfile);
			fclose(outfile);
		}
		else if (inputType == 5) {
			FILE* outfile = fopen(outfss.str().c_str(), "wb");
			saveOutputFile5(fname, outfile);
			fclose(outfile);
		}
		else if (inputType == 7) {
			FILE* outfile = fopen(outfss.str().c_str(), "wb");
			saveOutputFile7(fname, outfile);
			fclose(outfile);
		}
		else if (inputType == 8) {
			FILE* outfile = fopen(outfss.str().c_str(), "wb");
			saveOutputFile8(fname, outfile);
			fclose(outfile);
		}
		else {
			std::cout << "Input type not recognized" << std::endl;
			system("pause");
			return;
		}
		intervalfile << outfss.str() << std::endl;
	}

	std::cout << "Done. " << std::endl;
	system("pause");
}

//quality, 336 point intervals, sensor data
void saveOutputFile1(std::string infname, std::ofstream* outfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header

	while (std::getline(infile, line)) {
		std::replace(line.begin(), line.end(), ',', ' ');
		std::stringstream lss(line);
		std::string dum;
		lss >> dum;	//clipped flag
		if (dum == "True")
			continue;
		lss >> dum;	//peak mismatch flag
		if (dum == "True")
			continue;

		lss >> dum >> dum >> dum >> dum;	//SegmentNum, WindowNum, SegmentQuality, SegmentCorrelation

		float windowCorrelation;
		lss >> windowCorrelation;

		(*outfile) << "dummy," << windowCorrelation << ",";

		lss >> dum >> dum >> dum >> dum >> dum >> dum;	//SBP, DBP, 1_min_SBP, 1_min_DBP, SensorPeak, SensorValley

		float val;
		std::vector<float> vals;
		float minVal = 9999;
		float maxVal = -9999;
		for (size_t i = 0; i < SENSOR_LENGTH; i++) {
			lss >> val;
			vals.push_back(val);
			minVal = std::min(val, minVal);
			maxVal = std::max(val, maxVal);
		}
		for (size_t i = 0; i < vals.size(); i++) {
			(*outfile) << 2.0f*(vals[i] - minVal) / (maxVal - minVal) - 1.0f << ",";
		}
		(*outfile) << std::endl;
	}
}

//take 336 point interval from first 512 point interval
void saveOutputFile2(std::string infname, std::ofstream* outfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header

	while (std::getline(infile, line)) {
		std::replace(line.begin(), line.end(), ',', ' ');
		std::stringstream lss(line);
		std::string dum;

		lss >> dum >> dum;	//SegmentNum, WindowNum

		float SBP;
		float DBP;
		lss >> SBP >> DBP;

		(*outfile) << "dummy," << SBP << "," << DBP << ",";

		//88 dummy indices to accomodate the transition from 512 points to 336 points
		for (size_t i = 0; i < 512 / 2 - 336 / 2; i++)
			lss >> dum;

		float val;
		std::vector<float> vals;
		float minVal = 9999;
		float maxVal = -9999;
		for (size_t i = 0; i < SENSOR_LENGTH; i++) {
			lss >> val;
			vals.push_back(val);
			minVal = std::min(val, minVal);
			maxVal = std::max(val, maxVal);
		}
		for (size_t i = 0; i < vals.size(); i++) {
			(*outfile) << 2.0f*(vals[i] - minVal) / (maxVal - minVal) - 1.0f << ",";
		}
		(*outfile) << std::endl;
	}
}

//quality with 512 point intervals, sensor data, also includes inverted data
void saveOutputFile3(std::string infname, std::ofstream* outfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header

	while (std::getline(infile, line)) {
		std::replace(line.begin(), line.end(), ',', ' ');
		std::stringstream lss(line);
		std::string dum;
		lss >> dum;	//clipped flag
		if (dum == "True")
			continue;
		lss >> dum;	//peak mismatch flag
		if (dum == "True")
			continue;

		lss >> dum >> dum >> dum >> dum;	//SegmentNum, WindowNum, SegmentQuality, SegmentCorrelation

		float windowCorrelation;
		lss >> windowCorrelation;

		lss >> dum >> dum >> dum >> dum >> dum >> dum;	//SBP, DBP, 1_min_SBP, 1_min_DBP, SensorPeak, SensorValley

		float val;
		std::vector<float> vals;
		float minVal = 9999;
		float maxVal = -9999;
		for (size_t i = 0; i < 512; i++) {
			lss >> val;
			vals.push_back(val);
			minVal = std::min(val, minVal);
			maxVal = std::max(val, maxVal);
		}

		(*outfile) << "dummy," << windowCorrelation << ",";
		for (size_t i = 0; i < vals.size(); i++) {
			(*outfile) << 2.0f*(vals[i] - minVal) / (maxVal - minVal) - 1.0f << ",";
		}
		(*outfile) << std::endl;
		(*outfile) << "dummy," << -windowCorrelation << ",";
		for (size_t i = 0; i < vals.size(); i++) {
			(*outfile) << -(2.0f*(vals[i] - minVal) / (maxVal - minVal) - 1.0f) << ",";
		}
		(*outfile) << std::endl;
	}
}

//quality with 512 point intervals, sensor data, also includes inverted data; binary output
void saveOutputFile4(std::string infname, FILE* outbinfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header

	float dumVal = 0;
	fwrite(&dumVal, sizeof(float), 1, outbinfile);
	fwrite(&dumVal, sizeof(float), 1, outbinfile);	//dummy header

	while (std::getline(infile, line)) {
		std::replace(line.begin(), line.end(), ',', ' ');
		std::stringstream lss(line);
		std::string dum;
		lss >> dum;	//clipped flag
		if (dum == "True")
			continue;
		lss >> dum;	//peak mismatch flag
		if (dum == "True")
			continue;

		lss >> dum >> dum >> dum >> dum;	//SegmentNum, WindowNum, SegmentQuality, SegmentCorrelation

		float windowCorrelation;
		lss >> windowCorrelation;

		lss >> dum >> dum >> dum >> dum >> dum >> dum;	//SBP, DBP, 1_min_SBP, 1_min_DBP, SensorPeak, SensorValley

		float val;
		std::vector<float> vals;
		float minVal = 9999;
		float maxVal = -9999;
		for (size_t i = 0; i < 512; i++) {
			lss >> val;
			vals.push_back(val);
			minVal = std::min(val, minVal);
			maxVal = std::max(val, maxVal);
		}

		fwrite(&windowCorrelation, sizeof(float), 1, outbinfile);
		for (size_t i = 0; i < vals.size(); i++) {
			float val = 2.0f*(vals[i] - minVal) / (maxVal - minVal) - 1.0f;
			fwrite(&val, sizeof(float), 1, outbinfile);
		}
		windowCorrelation = -windowCorrelation;
		fwrite(&windowCorrelation, sizeof(float), 1, outbinfile);
		for (size_t i = 0; i < vals.size(); i++) {
			float val = -(2.0f*(vals[i] - minVal) / (maxVal - minVal) - 1.0f);
			fwrite(&val, sizeof(float), 1, outbinfile);
		}
	}
}

//Artline, 512 intervals, outputs: sBP, dBP, dummy, dummy, unscaled waveform
void saveOutputFile5(std::string infname, FILE* outbinfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header

	float dumVal = 0;
	fwrite(&dumVal, sizeof(float), 1, outbinfile);
	fwrite(&dumVal, sizeof(float), 1, outbinfile);	//dummy header

	while (std::getline(infile, line)) {
		std::replace(line.begin(), line.end(), ',', ' ');
		std::stringstream lss(line);
		std::string dum;

		lss >> dum >> dum;	//SegmentNum, WindowNum

		float sBP, dBP;
		lss >> sBP >> dBP;

		/*
		for (size_t i = 0; i < 512; i++)	//raw artline
			lss >> dum;
			*/

		float val;
		std::vector<float> vals;
		for (size_t i = 0; i < 512; i++) {
			lss >> val;
			vals.push_back(val);
		}

		fwrite(&dBP, sizeof(float), 1, outbinfile);
		fwrite(&sBP, sizeof(float), 1, outbinfile);
		float dumFloat = 0;
		fwrite(&dumFloat, sizeof(float), 1, outbinfile);
		fwrite(&dumFloat, sizeof(float), 1, outbinfile);
		for (size_t i = 0; i < vals.size(); i++)
			fwrite(&vals[i], sizeof(float), 1, outbinfile);
	}
}

//Sid fixed data
void saveOutputFile6(std::string infname, FILE* outbinfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header
}

//Meet sensor data with aline
void saveOutputFile7(std::string infname, FILE* outbinfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header

	float dumVal = 0;
	fwrite(&dumVal, sizeof(float), 1, outbinfile);
	fwrite(&dumVal, sizeof(float), 1, outbinfile);	//dummy header

	std::vector<float> outputs;
	std::vector<float> sensor(512);
	std::vector<float> peaks;
	size_t waveformId = 0;
	while (std::getline(infile, line)) {
		outputs.clear();
		outputs.resize(17);
		outputs[0] = waveformId;
		peaks.clear();
		peaks.resize(512);

		float sensorPeak = 0;
		float sensorValley = 0;
		
		size_t column = 0;
		//std::replace(line.begin(), line.end(), ',', ' ');
		std::stringstream lss(line);
		std::string dum;
		bool saveEvent = true;
		while (std::getline(lss, dum, ',')) {
			if (column == 0 && dum == "True") {
				saveEvent = false;
				break;
			}
			else if (column == 1 && dum == "True") {
				saveEvent = false;
				break;
			}
			else if (column == 4) {
				float qual;
				if(!(std::stringstream(dum) >> qual))
					std::cout << "Invalid column: " << dum << std::endl;
				if (fabs(qual) < 3) {
					saveEvent = false;
					break;
				}
			}
			else if (column == 7) {
				float sbp;
				if(!(std::stringstream(dum) >> sbp))
					std::cout << "Invalid column: " << dum << std::endl;
				outputs[2] = sbp;
			}
			else if (column == 9) {
				float dbp;
				if(!(std::stringstream(dum) >> dbp))
					std::cout << "Invalid column: " << dum << std::endl;
				outputs[1] = dbp;
			}
			else if (column == 13) {
				if(!(std::stringstream(dum) >> sensorPeak))
					std::cout << "Invalid column: " << dum << std::endl;
			}
			else if (column == 14) {
				if(!(std::stringstream(dum) >> sensorValley))
					std::cout << "Invalid column: " << dum << std::endl;
			}
			else if (column >= 15 && column <= 526) {
				if (!(std::stringstream(dum) >> sensor[column - 15]))
					std::cout << "Invalid column: " << dum << std::endl;
				if (sensorPeak > sensorValley)
					sensor[column - 15] = 2.0f*(sensor[column - 15] - sensorValley) / (sensorPeak - sensorValley) - 1.0f;
			}
			else if (column >= 527 && column <= 546) {
				size_t peakIndex;
				if ((std::stringstream(dum) >> peakIndex))
					peaks[peakIndex] = 1;
			}
			else if (column >= 547 && column <= 566) {
				size_t valleyIndex;
				if ((std::stringstream(dum) >> valleyIndex))
					peaks[valleyIndex] = -1;
			}

			column++;
		}
		if (saveEvent) {
			fwrite(&outputs[0], sizeof(float), outputs.size(), outbinfile);
			fwrite(&sensor[0], sizeof(float), sensor.size(), outbinfile);
			fwrite(&peaks[0], sizeof(float), peaks.size(), outbinfile);
		}
		waveformId++;
	}

}

//Meet sensor data with aline (earlier format, no peaks). Includes inverted waveform.
void saveOutputFile8(std::string infname, FILE* outbinfile) {
	std::ifstream infile(infname);

	std::string line;
	std::getline(infile, line);	//header

	float dumVal = 0;
	fwrite(&dumVal, sizeof(float), 1, outbinfile);
	fwrite(&dumVal, sizeof(float), 1, outbinfile);	//dummy header

	std::vector<float> outputs;
	std::vector<float> sensor(512);
	size_t waveformId = 0;
	while (std::getline(infile, line)) {
		outputs.clear();
		outputs.resize(2);
		
		float sensorPeak = 0;
		float sensorValley = 0;

		size_t column = 0;
		//std::replace(line.begin(), line.end(), ',', ' ');
		std::stringstream lss(line);
		std::string dum;
		bool saveEvent = true;

		while (std::getline(lss, dum, ',')) {
			if (column == 0 && dum == "True") {
				saveEvent = false;
				break;
			}
			else if (column == 1 && dum == "True") {
				saveEvent = false;
				break;
			}
			else if (column == 7) {
				float sbp;
				if(!(std::stringstream(dum) >> sbp))
					std::cout << "Invalid column: " << dum << std::endl;
				outputs[0] = sbp;
			}
			else if (column == 11) {
				if(!(std::stringstream(dum) >> sensorPeak))
					std::cout << "Invalid column: " << dum << std::endl;
			}
			else if (column == 12) {
				if(!(std::stringstream(dum) >> sensorValley))
					std::cout << "Invalid column: " << dum << std::endl;
			}
			else if (column >= 13 && column <= 524) {
				if (!(std::stringstream(dum) >> sensor[column - 13]))
					std::cout << "Invalid column: " << dum << std::endl;
				if (sensorPeak > sensorValley)
					sensor[column - 13] = 2.0f*(sensor[column - 13] - sensorValley) / (sensorPeak - sensorValley) - 1.0f;
				else
					sensor[column - 13] = 0;
			}

			column++;
		}
		if (saveEvent) {
			fwrite(&outputs[0], sizeof(float), outputs.size(), outbinfile);
			fwrite(&sensor[0], sizeof(float), sensor.size(), outbinfile);
			for (size_t s = 0; s < sensor.size(); s++) {
				sensor[s] = -sensor[s];
			}
			fwrite(&outputs[0], sizeof(float), outputs.size(), outbinfile);
			fwrite(&sensor[0], sizeof(float), sensor.size(), outbinfile);
		}
		waveformId++;
	}

}

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <list>

#define datastring ""
//#define datastring "D:/trivialNetworkTest/sethsensor/"

#define SAVE_PERIOD 10

#define COMMA_DELIMITER

struct SensorConvDatasetInfo {
	std::ifstream* datafile;
	size_t windowSize;
#ifndef SAVE_PERIOD
	size_t firstQualityColumn;
	float qualityThresh;
#endif
	size_t firstDataColumn;
	size_t lastDataColumn;
	std::string outputName;
	size_t outNum = 0;
};

struct IntervalData {
	std::string fname;
	std::string suffix;
	size_t startIndex;
	size_t endIndex;
	float output;
	SensorConvDatasetInfo* info;
};

void convertInterval(IntervalData* intData);
void convertTrainset(SensorConvDatasetInfo* info);

int main() {
	SensorConvDatasetInfo dataInfo;

	std::string datafname;
	std::cout << "Enter data file: ";
	std::cin >> datafname;

	std::ifstream datafile(datastring + datafname);
	if (!datafile.is_open()) {
		std::cout << "Can't find datafile" << std::endl;
		system("pause");
		return 0;
	}
	dataInfo.datafile = &datafile;

	std::cout << "Enter window size: ";
	std::cin >> dataInfo.windowSize;

	std::cout << "Enter first data column (starting at 0): ";
	std::cin >> dataInfo.firstDataColumn;
	
	std::cout << "Enter last data column: ";
	std::cin >> dataInfo.lastDataColumn;

#ifndef SAVE_PERIOD
	std::cout << "Enter quality threshold: ";
	std::cin >> dataInfo.qualityThresh;

	std::cout << "Enter first quality column: ";
	std::cin >> dataInfo.firstQualityColumn;
#endif

	std::cout << "Enter output file name: ";
	std::cin >> dataInfo.outputName;

	convertTrainset(&dataInfo);

	system("pause");
}

void convertTrainset(SensorConvDatasetInfo* info) {
	std::string line;

	while (std::getline((*info->datafile), line)) {
		IntervalData intData;
		std::stringstream lss(line);
		lss >> intData.fname >> intData.startIndex >> intData.endIndex >> intData.output;
		intData.info = info;
		if (!(lss >> intData.suffix))
			intData.suffix = "";
		else
			intData.suffix = "_" + intData.suffix;

		std::cout << "Reading interval: " << intData.fname << " " << intData.startIndex << " " << intData.endIndex << " " << intData.output << " " << intData.suffix << std::endl;
		info->outNum++;

		convertInterval(&intData);
	}
}

void convertInterval(IntervalData* intData) {
	std::ifstream infile(datastring + intData->fname);
	if (!infile.is_open()) {
		std::cout << "Couldn't open file " << datastring + intData->fname << std::endl;
		return;
	}
	std::stringstream outss;
	outss << datastring << intData->info->outputName << intData->suffix << "_" << intData->info->outNum;
	FILE* outfile = fopen(outss.str().c_str(), "wb");
	float header = 0;
	fwrite(&header, sizeof(float), 1, outfile);
	fwrite(&header, sizeof(float), 1, outfile);

	std::string line;
	for (size_t i = 0; i < intData->startIndex; i++) {
		if (infile.ignore(10000, infile.widen('\n'))){
			//just skipping the line
		}
		else {
			std::cout << "Some sort of error skipping initial lines of file " << std::endl;
			system("pause");
		}
	}

	std::vector<std::list<long long>> data;
	data.resize(intData->info->lastDataColumn - intData->info->firstDataColumn + 1);
	std::vector<float> quality;
	quality.resize(data.size());

	for (size_t i = intData->startIndex; i < intData->endIndex; i++) {
		std::getline(infile, line);
		std::stringstream lss(line);

		std::string dum;
		quality.clear();
#ifdef COMMA_DELIMITER
		for (size_t c = 0; std::getline(lss, dum, ','); c++)
#else
		for (size_t c = 0; lss >> dum; c++)
#endif
		{
			if (c >= intData->info->firstDataColumn && c <= intData->info->lastDataColumn) {
				long long val;
				(std::stringstream(dum)) >> val;
				size_t colNum = c - intData->info->firstDataColumn;
				data[colNum].push_back(val);
				if (data[colNum].size() > intData->info->windowSize)
					data[colNum].pop_front();
			}
#ifndef SAVE_PERIOD
			else if (c >= intData->info->firstQualityColumn) {
				float val;
				(std::stringstream(dum)) >> val;
				if (c - intData->info->firstQualityColumn < data.size())
					quality.push_back(val);
			}
#endif
		}

#ifndef SAVE_PERIOD
		for (size_t q = 0; q < quality.size(); q++) {
			if (fabs(quality[q]) >= intData->info->qualityThresh && data[q].size() == intData->info->windowSize)
#else
		for (size_t q = 0; q < data.size(); q++) {
			if ((i - intData->startIndex) % SAVE_PERIOD == 0 && data[q].size() == intData->info->windowSize)
#endif
			{
				for (size_t invert = 0; invert < 2; invert++) {
					std::vector<float> inputs;
					long long initVal = (invert == 0 ? data[q].front() : -data[q].front());
					long long maxVal = initVal;
					long long minVal = initVal;
					for (std::list<long long>::iterator it = data[q].begin(); it != data[q].end(); it++) {
						long long val;
						if (invert == 0)
							val = (*it);
						else
							val = -(*it);
						maxVal = std::max(maxVal, val);
						minVal = std::min(minVal, val);
					}

					for (std::list<long long>::iterator it = data[q].begin(); it != data[q].end(); it++) {
						long long val;
						if (invert == 0)
							val = (*it);
						else
							val = -(*it);

						long long valDiff = val - minVal;
						long long maxDiff = maxVal - minVal;

						if (maxDiff > 0)
							inputs.push_back(2.0f*valDiff / maxDiff - 1.0f);
						else
							inputs.push_back(0.0f);
					}

					fwrite(&intData->output, sizeof(float), 1, outfile);
#ifndef SAVE_PERIOD
					fwrite(&quality[q], sizeof(float), 1, outfile);
#else
					float dum = 0;
					fwrite(&dum, sizeof(float), 1, outfile);
#endif
					if (inputs.size() != intData->info->windowSize)
						std::cout << "Invalid size window!" << std::endl;
					else
						fwrite(&inputs[0], sizeof(float), inputs.size(), outfile);
				}
			}
		}
	}

	fclose(outfile);
}

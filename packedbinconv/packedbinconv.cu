
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <ctime>
#include <fstream>
#include <vector>
#include <sstream>

#define datastring "D:/momArtline/bindata/"

int main() {
	std::string infname;
	std::cout << "Enter name of input packed binary file: ";
	std::cin >> infname;

	FILE* infile = fopen((datastring + infname).c_str(), "rb");

	size_t version;
	fread(&version, sizeof(size_t), 1, infile);
	size_t columnNum;
	fread(&columnNum, sizeof(size_t), 1, infile);
	std::cout << "Version: " << version << " Num columns: " << columnNum << std::endl;

	std::vector<float> columns(columnNum);
	size_t readCount = 0;
	size_t entries = 0;
	do {
		readCount = fread(&columns[0], sizeof(float), columnNum, infile);
		for (size_t i = 0; i < columns.size(); i++)
			std::cout << columns[i] << " ";
		std::cout << std::endl;
		system("pause");
		if (readCount == columnNum)
			entries++;
	} while (readCount == columnNum);

	std::cout << entries << " entries read" << std::endl;
}
